#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#ifdef __INTELLISENSE__
void __syncthreads();
#endif

#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime_api.h>
#include <pgmio.h>
#include <vector>

// Block width WIDTH & HEIGHT
#define BLOCK_W 10
#define BLOCK_H 10

// prototype declarations

#define MAXLINE 128

float total, sobel;
hipEvent_t start_total, stop_total;
hipEvent_t start_sobel, stop_sobel;

__global__ void warm_up_gpu(){
  unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
  float ia, ib;
  ia = ib = 0.0f;
  ib += ia + tid; 
}


__global__ void imageBlur_horizontal(float *input, float *output, size_t width, size_t height) {

	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;

	int numcols = width;

	float blur;

	if (row <= height && col <= width && row > 0 && col > 0)
	{
		// weights
		int	x3, x4, x5;

		// blur
		// 0.2 0.2 0.2

		x3 = input[row * numcols + (col - 1)];			// left
		x4 = input[row * numcols + col];				// center
		x5 = input[row * numcols + (col + 1)];			// right

		blur =  (x3 * 0.2) + (x4 * 0.2) + (x5 * 0.2);

		output[row * numcols + col] = blur;
	}
}

__global__ void imageBlur_vertical(float *input, float *output, size_t width, size_t height) {
	
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;

	int numcols = width;

	float blur;

	if (row <= height && col <= width && row > 0 && col > 0)
	{
		// weights
		int		x1,x7;

		// blur
		// 0.0 0.2 0.0
		// 0.2 0.2 0.2
		// 0.0 0.2 0.0

		x1 = input[(row + 1) * numcols + col];			// up
		x7 = input[(row + -1) * numcols + col];			// down

		blur = (x1 * 0.2) + (x7 * 0.2);

		output[row * numcols + col] = blur;
	}
}

__global__ void gradient_horizontal(float *input, float *output, size_t width, size_t height) {

	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;

	int numcols = width;


	// horizontal
	// -1  0  1
	// -2  0  2
	// -1  0  1

	int x0, x2, 
		x3, x5, 
		x6, x8;

	if (row <= height && col <= width && row > 0 && col > 0)
	{
	
	x0 = input[(row - 1) * numcols + (col - 1)];	// leftup
	x2 = input[(row - 1) * numcols + (col + 1)];	// rightup
	x3 = input[row * numcols + (col - 1)];			// left
	x5 = input[row * numcols + (col + 1)];			// right
	x6 = input[(row + 1) * numcols + (col - 1)];	// leftdown
	x8 = input[(row + 1) * numcols + (col + 1)];	// rightdown


	output[row * numcols + col] = (x0 * -1) + (x2 * 1) + (x3 * -2) + (x5 * 2) + (x6 * -1) + (x8 * 1);

	}

	return;
}


__global__ void gradient_vertical(float *input, float *output, size_t width, size_t height) {

	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;

	int numcols = width;

	// vertical 
	// -1 -2 -1
	//  0  0  0
	//  1  2  1

	int x0, x1, 
		x3, x6, 
		x7, x8;

	if (row <= height && col <= width && row > 0 && col > 0)
	{
		x0 = input[(row - 1) * numcols + (col - 1)];	// leftup
		x1 = input[(row + 1) * numcols + col];			// up
		x3 = input[row * numcols + (col - 1)];			// left
		x6 = input[(row + 1) * numcols + (col - 1)];	// leftdown
		x7 = input[(row + -1) * numcols + col];			// down
		x8 = input[(row + 1) * numcols + (col + 1)];	// rightdown


		output[row * numcols + col] = (x0 * -1) + (x1 * -2) + (x3 * -1) + (x6 * 1) + (x7 * 2) + (x8 * 1);

	}
	
}

__global__ void sobelFilter(float *input, float *output, float *gradient_h_output, float *gradient_v_output, size_t width, size_t height) {
	
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;

	int numcols = width;

	float gradient,gradient_h,gradient_v;	
	float thresh = 30;	

	if (row <= height && col <= width && row > 0 && col > 0)
	{	

		gradient_h = gradient_h_output[row * numcols + col];
		gradient_v = gradient_v_output[row * numcols + col];
		gradient = sqrt((gradient_h * gradient_h) + (gradient_v * gradient_v));

		if (gradient >= thresh)
		{
			gradient = 255;
		}
		else {
			gradient = 0;
		}
		output[row * numcols + col] = gradient;
	}
}

int main(int argc, char *argv[])
{
	size_t width = 600, height = 600;
	int devID = findCudaDevice(argc, (const char **)argv);
	float *image = NULL, *final = NULL;
	float *image2 = NULL, *final2 = NULL;
	size_t memSize = width * height * sizeof(float);
	checkCudaErrors((hipHostMalloc(&image, memSize)));
	checkCudaErrors((hipHostMalloc(&image2, memSize)));
	checkCudaErrors((hipHostMalloc(&final, memSize)));
	checkCudaErrors((hipHostMalloc(&final2, memSize)));

	// read image 
	pgmread("../images/test_images/apollonian_gasket.ascii.pgm", (void *)image, width, height);
	pgmread("../images/test_images/apollonian_gasket.ascii.pgm", (void *)image2, width, height);

	hipEventCreate(&start_total);
	hipEventCreate(&stop_total);
	hipEventRecord(start_total, 0);

	int x, y;
	float *d_input, *d_output, *gradient_h_output, *gradient_v_output;
	float *d_input2, *d_output2, *gradient_h_output2, *gradient_v_output2;

	printf("Block size: %dx%d\n", BLOCK_W, BLOCK_H);

	checkCudaErrors(hipMalloc(&d_input, memSize));
	checkCudaErrors(hipMalloc(&d_output, memSize));
	checkCudaErrors(hipMalloc(&gradient_h_output, memSize));
	checkCudaErrors(hipMalloc(&gradient_v_output, memSize));

	checkCudaErrors(hipMalloc(&d_input2, memSize));
	checkCudaErrors(hipMalloc(&d_output2, memSize));
	checkCudaErrors(hipMalloc(&gradient_h_output2, memSize));
	checkCudaErrors(hipMalloc(&gradient_v_output2, memSize));

	printf("Blocks per grid (width): %d |", (width / BLOCK_W));
	printf("Blocks per grid (height): %d \n", (height / BLOCK_H));

  	hipGraph_t graph;
  	std::vector<hipGraphNode_t> nodeDependencies;
	std::vector<hipGraphNode_t> nodeDependencies2;

	checkCudaErrors(hipGraphCreate(&graph, 0));

	
	hipGraphNode_t memcpyNode;
	hipMemcpy3DParms memcpyParams = {0};

	memcpyParams.srcArray = NULL;
  	memcpyParams.srcPos = make_hipPos(0, 0, 0);
  	memcpyParams.srcPtr =
      make_hipPitchedPtr(image, memSize, 1, 1);
  	memcpyParams.dstArray = NULL;
  	memcpyParams.dstPos = make_hipPos(0, 0, 0);
  	memcpyParams.dstPtr =
      make_hipPitchedPtr(d_input, memSize, 1, 1);
  	memcpyParams.extent = make_hipExtent(memSize, 1, 1);
  	memcpyParams.kind = hipMemcpyHostToDevice;

	checkCudaErrors(
      hipGraphAddMemcpyNode(&memcpyNode, graph, NULL, 0, &memcpyParams));
	nodeDependencies.push_back(memcpyNode);

	hipMemcpy3DParms memcpyParams01 = {0};

	memcpyParams01.srcArray = NULL;
  	memcpyParams01.srcPos = make_hipPos(0, 0, 0);
  	memcpyParams01.srcPtr =
      make_hipPitchedPtr(image2, memSize, 1, 1);
  	memcpyParams01.dstArray = NULL;
  	memcpyParams01.dstPos = make_hipPos(0, 0, 0);
  	memcpyParams01.dstPtr =
      make_hipPitchedPtr(d_input2, memSize, 1, 1);
  	memcpyParams01.extent = make_hipExtent(memSize, 1, 1);
  	memcpyParams01.kind = hipMemcpyHostToDevice;

	hipGraphNode_t memcpyNode2;
	checkCudaErrors(
      hipGraphAddMemcpyNode(&memcpyNode2, graph, NULL, 0, &memcpyParams01));
	nodeDependencies2.push_back(memcpyNode2);

	dim3 threads(BLOCK_W, BLOCK_H); // threads per block
	dim3 blocks(width / BLOCK_W, height / BLOCK_H); // blocks per grid 

	hipGraphNode_t kernelNode;
	hipKernelNodeParams kernelNodeParams = {0};

	void* kernelArgs0[4] = {(void *)&d_input,(void *)&d_output, &width, &height};
	kernelNodeParams.func = (void *)imageBlur_horizontal;
 	kernelNodeParams.gridDim = blocks;
  	kernelNodeParams.blockDim = threads;
  	kernelNodeParams.sharedMemBytes = 0;
 	kernelNodeParams.kernelParams = (void **)kernelArgs0;
 	kernelNodeParams.extra = NULL;


	checkCudaErrors(
    hipGraphAddKernelNode(&kernelNode, graph, &memcpyNode,
                             1, &kernelNodeParams));

  	nodeDependencies.clear();
  	nodeDependencies.push_back(kernelNode);
  
	hipGraphNode_t kernelNode2;
	hipKernelNodeParams kernelNodeParams01 = {0};

	void* kernelArgs1[4] = {(void *)&d_input2,(void *)&d_output2, &width, &height};
	kernelNodeParams01.func = (void *)imageBlur_horizontal;
 	kernelNodeParams01.gridDim = blocks;
  	kernelNodeParams01.blockDim = threads;
  	kernelNodeParams01.sharedMemBytes = 0;
 	kernelNodeParams01.kernelParams = (void **)kernelArgs1;
 	kernelNodeParams01.extra = NULL;

	checkCudaErrors(
    hipGraphAddKernelNode(&kernelNode2, graph, &memcpyNode2,
                             1, &kernelNodeParams));
  	nodeDependencies2.clear();
  	nodeDependencies2.push_back(kernelNode2);
	
	kernelNodeParams.func = (void *)imageBlur_vertical;

	checkCudaErrors(
    hipGraphAddKernelNode(&kernelNode, graph, &memcpyNode,
                             1, &kernelNodeParams));

  	nodeDependencies.push_back(kernelNode);

	kernelNodeParams01.func = (void *)imageBlur_vertical;
	
	checkCudaErrors(
    hipGraphAddKernelNode(&kernelNode2, graph, &memcpyNode2,
                             1, &kernelNodeParams01));

  	nodeDependencies2.push_back(kernelNode2);

	hipGraphNode_t empty_node;
	checkCudaErrors(
      hipGraphAddEmptyNode(&empty_node, graph, nodeDependencies.data(),
                             nodeDependencies.size()));

	hipGraphNode_t empty_node2;
	checkCudaErrors(
      hipGraphAddEmptyNode(&empty_node2, graph, nodeDependencies2.data(),
                             nodeDependencies2.size()));

	void* kernelArgs2[4] = {(void *)&d_input, (void *)&gradient_h_output, &width, &height};
	kernelNodeParams.func = (void *)gradient_horizontal;
 	kernelNodeParams.kernelParams = (void **)kernelArgs2;
	
	checkCudaErrors(
    hipGraphAddKernelNode(&kernelNode, graph, &empty_node,
                             1, &kernelNodeParams));

  	nodeDependencies.clear();
  	nodeDependencies.push_back(kernelNode);

	void* kernelArgs21[4] = {(void *)&d_input2, (void *)&gradient_h_output2, &width, &height};
	kernelNodeParams01.func = (void *)gradient_horizontal;
 	kernelNodeParams01.kernelParams = (void **)kernelArgs21;
	
	checkCudaErrors(
    hipGraphAddKernelNode(&kernelNode2, graph, &empty_node2,
                             1, &kernelNodeParams01));

  	nodeDependencies2.clear();
  	nodeDependencies2.push_back(kernelNode2);
	
	void* kernelArgs3[4] = {(void *)&d_input,(void *)& gradient_v_output, &width, &height};
	kernelNodeParams.func = (void *)gradient_vertical;
 	kernelNodeParams.kernelParams = (void **)kernelArgs3;

	checkCudaErrors(
    hipGraphAddKernelNode(&kernelNode, graph, &empty_node,
                             1, &kernelNodeParams));

  	nodeDependencies.push_back(kernelNode);

	void* kernelArgs31[4] = {(void *)&d_input2,(void *)& gradient_v_output2, &width, &height};
	kernelNodeParams.func = (void *)gradient_vertical;
 	kernelNodeParams.kernelParams = (void **)kernelArgs31;

	checkCudaErrors(
    hipGraphAddKernelNode(&kernelNode2, graph, &empty_node2,
                             1, &kernelNodeParams01));

  	nodeDependencies2.push_back(kernelNode2);

	void* kernelArgs4[6] = {(void *)&d_input, (void *)&d_output, (void *)&gradient_h_output, (void *)&gradient_v_output, &width, &height};
	kernelNodeParams.func = (void *)sobelFilter;
 	kernelNodeParams.kernelParams = (void **)kernelArgs4;

	checkCudaErrors(
    hipGraphAddKernelNode(&kernelNode, graph, nodeDependencies.data(),
                             nodeDependencies.size(), &kernelNodeParams));

  	nodeDependencies.clear();
  	nodeDependencies.push_back(kernelNode);

	void* kernelArgs41[6] = {(void *)&d_input2, (void *)&d_output2, (void *)&gradient_h_output2, (void *)&gradient_v_output2, &width, &height};
	kernelNodeParams01.func = (void *)sobelFilter;
 	kernelNodeParams01.kernelParams = (void **)kernelArgs41;

	checkCudaErrors(
    hipGraphAddKernelNode(&kernelNode2, graph, nodeDependencies2.data(),
                             nodeDependencies2.size(), &kernelNodeParams01));

  	nodeDependencies2.clear();
  	nodeDependencies2.push_back(kernelNode2);

	memcpyParams.srcPtr = make_hipPitchedPtr(d_output, memSize, 1, 1);
	memcpyParams.dstPtr = make_hipPitchedPtr(final, memSize, 1, 1);
	memcpyParams.extent = make_hipExtent(memSize, 1, 1);
	memcpyParams.kind = hipMemcpyDeviceToHost;

	checkCudaErrors(
      hipGraphAddMemcpyNode(&memcpyNode, graph, &kernelNode,
                             1, &memcpyParams));


	memcpyParams01.srcPtr = make_hipPitchedPtr(d_output2, memSize, 1, 1);
	memcpyParams01.dstPtr = make_hipPitchedPtr(final2, memSize, 1, 1);
	memcpyParams01.extent = make_hipExtent(memSize, 1, 1);
	memcpyParams01.kind = hipMemcpyDeviceToHost;

	checkCudaErrors(
      hipGraphAddMemcpyNode(&memcpyNode2, graph, &kernelNode2,
                             1, &memcpyParams01));

	checkCudaErrors(hipGraphDebugDotPrint(graph, "mainGraph.dot", 0));


	hipGraphExec_t graphExec;
  	checkCudaErrors(hipGraphInstantiate(&graphExec, graph, NULL, NULL, 0));
	
	warm_up_gpu << <blocks, threads >> > ();

	hipEventCreate(&start_sobel);
  	hipEventCreate(&stop_sobel);

    hipEventRecord(start_sobel, 0);
	
	checkCudaErrors(hipGraphLaunch(graphExec, 0));
	checkCudaErrors(hipStreamSynchronize(0));

	hipEventRecord(stop_sobel, 0);
  	hipEventSynchronize(stop_sobel);
  	hipEventElapsedTime(&sobel, start_sobel, stop_sobel);

	printf("Device Time:  %f s \n", sobel/1000);

	hipError_t err = hipGetLastError();
	if (hipSuccess != err)
	{
		fprintf(stderr, "Cuda error: %s: %s.\n", "Main Loop", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	checkCudaErrors(hipGraphExecDestroy(graphExec));
  	checkCudaErrors(hipGraphDestroy(graph));
	hipFree(d_input);
	hipFree(d_output);
	hipFree(gradient_h_output);
	hipFree(gradient_v_output);

	hipFree(d_input2);
	hipFree(d_output2);
	hipFree(gradient_h_output2);
	hipFree(gradient_v_output2);


   
  hipEventRecord(stop_total, 0);
  hipEventSynchronize(stop_total);
  hipEventElapsedTime(&total, start_total, stop_total);

  printf("Total Time:  %f s \n", total/1000);
  
	// write image
	pgmwrite("../images/image-output_g_apollonian_gasket.ascii.pgm", (void *)final,width, height);
	pgmwrite("../images/image-output2_g_apollonian_gasket.ascii.pgm", (void *)final2,width, height);
	hipDeviceReset();
	
	return 0;


}

