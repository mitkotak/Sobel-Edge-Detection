#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#ifdef __INTELLISENSE__
void __syncthreads();
#endif

#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>
#include <pgmio.h>
#include <vector>

// image dimensions WIDTH & HEIGHT
#define NIMAGES 2048

// Block width WIDTH & HEIGHT
#define BLOCK_W 10
#define BLOCK_H 10

// buffer to read image into
// float image[HEIGHT][WIDTH];

// buffer for resulting image
// float final[HEIGHT][WIDTH];


#define MAXLINE 128

float total, sobel;
hipEvent_t start_total, stop_total;
hipEvent_t start_sobel, stop_sobel;

__global__ void warm_up_gpu(){
  unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
  float ia, ib;
  ia = ib = 0.0f;
  ib += ia + tid; 
}

__global__ void imageBlur_horizontal(float *input, float *output, int width, int height) {

	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;

	int numcols = width;

	float blur;

	if (row <= height && col <= width && row > 0 && col > 0)
	{
		// weights
		int	x3, x4, x5;

		// blur
		// 0.2 0.2 0.2

		x3 = input[row * numcols + (col - 1)];			// left
		x4 = input[row * numcols + col];				// center
		x5 = input[row * numcols + (col + 1)];			// right

		blur =  (x3 * 0.2) + (x4 * 0.2) + (x5 * 0.2);

		output[row * numcols + col] = blur;
	}
}

__global__ void imageBlur_vertical(float *input, float *output, int width, int height) {
	
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;

	int numcols = width;

	float blur;

	if (row <= height && col <= width && row > 0 && col > 0)
	{
		// weights
		int		x1,x7;

		// blur
		// 0.0 0.2 0.0
		// 0.2 0.2 0.2
		// 0.0 0.2 0.0

		x1 = input[(row + 1) * numcols + col];			// up
		x7 = input[(row + -1) * numcols + col];			// down

		blur = (x1 * 0.2) + (x7 * 0.2);

		output[row * numcols + col] = blur;
	}
}


__global__ void gradient_horizontal(float *input, float *output, int width, int height) {

	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;

	int numcols = width;


	// horizontal
	// -1  0  1
	// -2  0  2
	// -1  0  1

	int x0, x2, 
		x3, x5, 
		x6, x8;

	if (row <= height && col <= width && row > 0 && col > 0)
	{
	
	x0 = input[(row - 1) * numcols + (col - 1)];	// leftup
	x2 = input[(row - 1) * numcols + (col + 1)];	// rightup
	x3 = input[row * numcols + (col - 1)];			// left
	x5 = input[row * numcols + (col + 1)];			// right
	x6 = input[(row + 1) * numcols + (col - 1)];	// leftdown
	x8 = input[(row + 1) * numcols + (col + 1)];	// rightdown


	output[row * numcols + col] = (x0 * -1) + (x2 * 1) + (x3 * -2) + (x5 * 2) + (x6 * -1) + (x8 * 1);

	}

	return;
}


__global__ void gradient_vertical(float *input, float *output, int width, int height) {

	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;

	int numcols = width;

	// vertical 
	// -1 -2 -1
	//  0  0  0
	//  1  2  1

	int x0, x1, 
		x3, x6, 
		x7, x8;

	if (row <= height && col <= width && row > 0 && col > 0)
	{
		x0 = input[(row - 1) * numcols + (col - 1)];	// leftup
		x1 = input[(row + 1) * numcols + col];			// up
		x3 = input[row * numcols + (col - 1)];			// left
		x6 = input[(row + 1) * numcols + (col - 1)];	// leftdown
		x7 = input[(row + -1) * numcols + col];			// down
		x8 = input[(row + 1) * numcols + (col + 1)];	// rightdown


		output[row * numcols + col] = (x0 * -1) + (x1 * -2) + (x3 * -1) + (x6 * 1) + (x7 * 2) + (x8 * 1);

	}
	
}

__global__ void sobelFilter(float *input, float *output, float *gradient_h_output, float *gradient_v_output, int width, int height) {
	
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;

	int numcols = width;

	float gradient,gradient_h,gradient_v;	
	float thresh = 30;	

	if (row <= height && col <= width && row > 0 && col > 0)
	{	

		gradient_h = gradient_h_output[row * numcols + col];
		gradient_v = gradient_v_output[row * numcols + col];
		gradient = sqrt((gradient_h * gradient_h) + (gradient_v * gradient_v));

		if (gradient >= thresh)
		{
			gradient = 255;
		}
		else {
			gradient = 0;
		}
		output[row * numcols + col] = gradient;
	}
}

	
	// pgmread("image100000x100000.pgm", (void *)image, WIDTH, HEIGHT);
	// pgmread("../images/image20000x20000.pgm", (void *)image, WIDTH, HEIGHT);
	// pgmread("../images/image16384x16384.pgm", (void *)image, WIDTH, HEIGHT);
	// pgmread("image10000x10000.pgm", (void *)image, WIDTH, HEIGHT);
	// pgmread("../images/image4096x4096.pgm", (void *)image, WIDTH, HEIGHT);
	// pgmread("../images/image2048x2048.pgm", (void *)image, WIDTH, HEIGHT);
	// pgmread("../images/rabbit2000x3000.pgm", (void *)image, WIDTH, HEIGHT);
	// pgmwrite("../images/image1024x1024.pgm", (void *)image, WIDTH, HEIGHT);
	// pgmread("../images/image512x512.pgm", (void *)image, WIDTH, HEIGHT);
	// pgmread("pgmimg.pgm", (void *)image, WIDTH, HEIGHT);

	// pgmwrite("image-outputl100000x100000.pgm", (void *)final, WIDTH, HEIGHT);
	// pgmwrite("image-output_ng_20000x20000.pgm", (void *)final, WIDTH, HEIGHT);
	// pgmwrite("../images/image-output_ng_16384x16384.pgm", (void *)final, WIDTH, HEIGHT);
	// pgmwrite("image-outputl10000x10000.pgm", (void *)final, WIDTH, HEIGHT);
	// pgmwrite("../images/image-output_g_4096x4096.pgm", (void *)final, WIDTH, HEIGHT);
	// pgmwrite("../images/image-output_ng_2048x2048.pgm", (void *)final, WIDTH, HEIGHT);
	// pgmwrite("../images/image-output_ng_2000x3000.pgm", (void *)final, WIDTH, HEIGHT);
	// pgmwrite("../images/image-output_ng_1024x1024.pgm", (void *)final, WIDTH, HEIGHT);
	// pgmwrite("../images/image-output_ng_512x512.pgm", (void *)final, WIDTH, HEIGHT);
	// // pgmwrite("pgmimg-output.pgm", (void *)final, WIDTH, HEIGHT);

int main(int argc, char *argv[])
{ 	int width = 600, height=600;

	size_t memSize = width * height * sizeof(float);
	float *image_array[NIMAGES];
	float *final_array[NIMAGES];
	float *d_input_array[NIMAGES];
	float *d_output_array[NIMAGES];
	float *gradient_h_output_array[NIMAGES];
	float *gradient_v_output_array[NIMAGES];

	for (int i=1; i <= NIMAGES; i++){
		float *image = NULL;
		checkCudaErrors((hipHostMalloc(&image, memSize)));
		pgmread("../images/test_images/apollonian_gasket.ascii.pgm", (void *)image, width, height);
		image_array[i] = image;
		
		float *final = NULL;
		checkCudaErrors((hipHostMalloc(&final, memSize)));
		final_array[i] = final;
		
		float *d_input, *d_output, *gradient_h_output, *gradient_v_output;

		checkCudaErrors(hipMalloc(&d_input, memSize));
		checkCudaErrors(hipMalloc(&d_output, memSize));
		checkCudaErrors(hipMalloc(&gradient_h_output, memSize));
		checkCudaErrors(hipMalloc(&gradient_v_output, memSize));

		d_input_array[i] = d_input;
		d_output_array[i] = d_output;
		gradient_h_output_array[i] = gradient_h_output;
		gradient_v_output_array[i] = gradient_v_output;

	}

  hipEventCreate(&start_total);
  hipEventCreate(&stop_total);
  hipEventRecord(start_total, 0);


	printf("Block size: %dx%d\n", BLOCK_W, BLOCK_H);

	printf("Blocks per grid (width): %d |", (width / BLOCK_W));
	printf("Blocks per grid (height): %d \n", (height / BLOCK_H));
	
	dim3 threads(BLOCK_W, BLOCK_H); // threads per block
	dim3 blocks(width / BLOCK_W, height / BLOCK_H); // blocks per grid 

	warm_up_gpu << <blocks, threads >> > ();

	float avg_sobel = 0.0;

	for (int j=1; j <= 20; j++){
	
	hipEventCreate(&start_sobel);
  	hipEventCreate(&stop_sobel);

	hipEventRecord(start_sobel, 0);

	for (int i=1; i<=NIMAGES; i++){

		float *image = image_array[i];
		float *d_input = d_input_array[i];
		float *d_output = d_output_array[i];
		float *gradient_h_output = gradient_h_output_array[i];
		float *gradient_v_output = gradient_v_output_array[i];
		float* final = final_array[i];

	hipMemcpy(d_input, image, memSize, hipMemcpyHostToDevice);

	// printf("Launching imageBlur_horizontal \n");
  	imageBlur_horizontal << <blocks, threads >> > (d_input, d_output, width, height);
	// printf("Launching imageBlur_vertical \n");
	imageBlur_vertical << <blocks, threads >> > (d_input, d_output, width, height);

	// printf("Copying data to device \n");
	// printf("Launching gradient_horizontal \n");
	gradient_horizontal<< <blocks, threads>> >(d_input, gradient_h_output, width, height);
	// printf("Launching gradient_vertical \n");
	gradient_vertical<< <blocks, threads>> >(d_input, gradient_v_output, width, height);
	// printf("Launching sobelFilter \n");	
	sobelFilter << <blocks, threads >> > (d_input, d_output, gradient_h_output, gradient_v_output, width, height);

	// printf("Copying data back to host \n");
	hipMemcpy(final, d_output, memSize, hipMemcpyDeviceToHost);

	}
	
	hipDeviceSynchronize();

	hipEventRecord(stop_sobel, 0);
  	hipEventSynchronize(stop_sobel);
  	hipEventElapsedTime(&sobel, start_sobel, stop_sobel);
	
	 avg_sobel += sobel/20;

	}

	printf("Total Avg Device Time:  %f s \n", avg_sobel/1000);

	hipError_t err = hipGetLastError();
	if (hipSuccess != err)
	{
		fprintf(stderr, "Cuda error: %s: %s.\n", "Main Loop", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	for (int i=1; i <= NIMAGES; i++){
		float *d_input = d_input_array[i];
		hipFree(d_input);
		float *d_output = d_output_array[i];
		hipFree(d_output);
		float *gradient_h_output = gradient_h_output_array[i];
		hipFree(gradient_h_output);
		float *gradient_v_output = gradient_v_output_array[i];
		hipFree(gradient_v_output);

		float* final = final_array[i];
		// write image
		pgmwrite("../images/test_images/image-output_g_apollonian_gasket.ascii.pgm", (void *)final,width, height);
	}
 
  hipEventRecord(stop_total, 0);
  hipEventSynchronize(stop_total);
  hipEventElapsedTime(&total, start_total, stop_total);

  printf("Total Time:  %f s \n", total/1000);
    
	hipDeviceReset();
	
	return 0;
}

