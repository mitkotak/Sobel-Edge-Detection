#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#ifdef __INTELLISENSE__
void __syncthreads();
#endif

#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>
#include <pgmio.h>

// image dimensions WIDTH & HEIGHT
#define WIDTH 256
#define HEIGHT 256

// Block width WIDTH & HEIGHT
#define BLOCK_W 16
#define BLOCK_H 16

// buffer to read image into
float image[HEIGHT][WIDTH];

// buffer for resulting image
float final[HEIGHT][WIDTH];

// prototype declarations
void load_image();
void call_kernel();
void save_image();

#define MAXLINE 128

float total, sobel;
hipEvent_t start_total, stop_total;
hipEvent_t start_sobel, stop_sobel;


__global__ void imageBlur(float *input, float *output, int width, int height) {
	
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;

	int numcols = WIDTH;

	float blur;

	if (row <= height && col <= width && row > 0 && col > 0)
	{
		// weights
		int		x1,
			x3, x4, x5,
				x7;

		// blur
		// 0.0 0.2 0.0
		// 0.2 0.2 0.2
		// 0.0 0.2 0.0

		x1 = input[(row + 1) * numcols + col];			// up
		x3 = input[row * numcols + (col - 1)];			// left
		x4 = input[row * numcols + col];				// center
		x5 = input[row * numcols + (col + 1)];			// right
		x7 = input[(row + -1) * numcols + col];			// down

		blur = (x1 * 0.2) + (x3 * 0.2) + (x4 * 0.2) + (x5 * 0.2) + (x7 * 0.2);

		output[row * numcols + col] = blur;
	}
}


__global__ void sobelFilter(float *input, float *output, int width, int height) {
	
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;

	int numcols = WIDTH;

	float gradient_h;	
	float gradient_v;	
	float gradient;		
	float thresh = 30;	

	if (row <= height && col <= width && row > 0 && col > 0)
	{
		int x0, x1, x2, 
		    x3,	    x5, 
			x6, x7, x8;

		// horizontal
		// -1  0  1
		// -2  0  2
		// -1  0  1

		// vertical 
		// -1 -2 -1
		//  0  0  0
		//  1  2  1

		x0 = input[(row - 1) * numcols + (col - 1)];	// leftup
		x1 = input[(row + 1) * numcols + col];			// up
		x2 = input[(row - 1) * numcols + (col + 1)];	// rightup
		x3 = input[row * numcols + (col - 1)];			// left
		x5 = input[row * numcols + (col + 1)];			// right
		x6 = input[(row + 1) * numcols + (col - 1)];	// leftdown
		x7 = input[(row + -1) * numcols + col];			// down
		x8 = input[(row + 1) * numcols + (col + 1)];	// rightdown


		gradient_h = (x0 * -1) + (x2 * 1) + (x3 * -2) + (x5 * 2) + (x6 * -1) + (x8 * 1);
		gradient_v = (x0 * -1) + (x1 * -2) + (x3 * -1) + (x6 * 1) + (x7 * 2) + (x8 * 1);

		gradient = sqrt((gradient_h * gradient_h) + (gradient_v * gradient_v));

		if (gradient >= thresh)
		{
			gradient = 255;
		}
		else {
			gradient = 0;
		}
		output[row * numcols + col] = gradient;
	}
}

void load_image() {
	pgmread("image512x512.pgm", (void *)image, WIDTH, HEIGHT);
}

void save_image() {
	pgmwrite("image-outputl512x512.pgm", (void *)final, WIDTH, HEIGHT);
}

void call_kernel() {
	int x, y;
	float *d_input, *d_output;

	printf("Block size: %dx%d\n", BLOCK_W, BLOCK_H);

	size_t memSize = WIDTH * HEIGHT * sizeof(float);

	hipMalloc(&d_input, memSize);
	hipMalloc(&d_output, memSize);

	for (y = 0; y < HEIGHT; y++) {
		for (x = 0; x < WIDTH; x++) {
			final[x][y] = 0.0;
		}
	}

	printf("Blocks per grid (width): %d |", (WIDTH / BLOCK_W));
	printf("Blocks per grid (height): %d |", (HEIGHT / BLOCK_H));


	hipMemcpy(d_input, image, memSize, hipMemcpyHostToDevice);

	dim3 threads(BLOCK_W, BLOCK_H); // threads per block
	dim3 blocks(WIDTH / BLOCK_W, HEIGHT / BLOCK_H); // blocks per grid 
  
  	imageBlur << <blocks, threads >> > (d_input, d_output, WIDTH, HEIGHT);
  
  	hipDeviceSynchronize();
    
  	hipMemcpy(d_input, d_output, memSize, hipMemcpyDeviceToHost);

	sobelFilter << <blocks, threads >> > (d_input, d_output, WIDTH, HEIGHT);

	hipDeviceSynchronize();

	hipMemcpy(final, d_output, memSize, hipMemcpyDeviceToHost);


	hipError_t err = hipGetLastError();
	if (hipSuccess != err)
	{
		fprintf(stderr, "Cuda error: %s: %s.\n", "Main Loop", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	hipFree(d_input);
	hipFree(d_output);
}

int main(int argc, char *argv[])
{
  hipEventCreate(&start_total);
  hipEventCreate(&stop_total);
    
  hipEventCreate(&start_sobel);
  hipEventCreate(&stop_sobel);
    
  hipEventRecord(start_total, 0);

	load_image();
   
  hipEventRecord(start_sobel, 0);

	call_kernel();
  
  hipEventRecord(stop_sobel, 0);
  hipEventSynchronize(stop_sobel);
  hipEventElapsedTime(&sobel, start_sobel, stop_sobel);

	save_image();
   
  hipEventRecord(stop_total, 0);
  hipEventSynchronize(stop_total);
  hipEventElapsedTime(&total, start_total, stop_total);
    
  printf("Total Parallel Time:  %f s |", sobel/1000);
  printf("Total Serial Time:  %f s |", (total-sobel)/1000);
  printf("Total Time:  %f s |", total/1000);
  
    
	hipDeviceReset();
	
	return 0;
}

