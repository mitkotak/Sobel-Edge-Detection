#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#ifdef __INTELLISENSE__
void __syncthreads();
#endif

#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>
#include <pgmio.h>
#include <vector>

// image dimensions WIDTH & HEIGHT
#define WIDTH 256
#define HEIGHT 256

// Block width WIDTH & HEIGHT
#define BLOCK_W 16
#define BLOCK_H 16

// buffer to read image into
float image[HEIGHT][WIDTH];

// buffer for resulting image
float final[HEIGHT][WIDTH];

// prototype declarations

void load_image();
void call_kernel();
void save_image();

#define MAXLINE 128

float total, sobel;
hipEvent_t start_total, stop_total;
hipEvent_t start_sobel, stop_sobel;


__global__ void imageBlur_horizontal(float *input, float *output, size_t width, size_t height) {

	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;

	int numcols = WIDTH;

	float blur;

	if (row <= height && col <= width && row > 0 && col > 0)
	{
		// weights
		int	x3, x4, x5;

		// blur
		// 0.2 0.2 0.2

		x3 = input[row * numcols + (col - 1)];			// left
		x4 = input[row * numcols + col];				// center
		x5 = input[row * numcols + (col + 1)];			// right

		blur =  (x3 * 0.2) + (x4 * 0.2) + (x5 * 0.2);

		output[row * numcols + col] = blur;
	}
}

__global__ void imageBlur_vertical(float *input, float *output, size_t width, size_t height) {
	
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;

	int numcols = WIDTH;

	float blur;

	if (row <= height && col <= width && row > 0 && col > 0)
	{
		// weights
		int		x1,x7;

		// blur
		// 0.0 0.2 0.0
		// 0.2 0.2 0.2
		// 0.0 0.2 0.0

		x1 = input[(row + 1) * numcols + col];			// up
		x7 = input[(row + -1) * numcols + col];			// down

		blur = (x1 * 0.2) + (x7 * 0.2);

		output[row * numcols + col] = blur;
	}
}

__global__ void gradient_horizontal(float *input, float *output, size_t width, size_t height) {

	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;

	int numcols = WIDTH;


	// horizontal
	// -1  0  1
	// -2  0  2
	// -1  0  1

	int x0, x2, 
		x3, x5, 
		x6, x8;

	if (row <= height && col <= width && row > 0 && col > 0)
	{
	
	x0 = input[(row - 1) * numcols + (col - 1)];	// leftup
	x2 = input[(row - 1) * numcols + (col + 1)];	// rightup
	x3 = input[row * numcols + (col - 1)];			// left
	x5 = input[row * numcols + (col + 1)];			// right
	x6 = input[(row + 1) * numcols + (col - 1)];	// leftdown
	x8 = input[(row + 1) * numcols + (col + 1)];	// rightdown


	output[row * numcols + col] = (x0 * -1) + (x2 * 1) + (x3 * -2) + (x5 * 2) + (x6 * -1) + (x8 * 1);

	}

	return;
}


__global__ void gradient_vertical(float *input, float *output, size_t width, size_t height) {

	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;

	int numcols = WIDTH;

	// vertical 
	// -1 -2 -1
	//  0  0  0
	//  1  2  1

	int x0, x1, 
		x3, x6, 
		x7, x8;

	if (row <= height && col <= width && row > 0 && col > 0)
	{
		x0 = input[(row - 1) * numcols + (col - 1)];	// leftup
		x1 = input[(row + 1) * numcols + col];			// up
		x3 = input[row * numcols + (col - 1)];			// left
		x6 = input[(row + 1) * numcols + (col - 1)];	// leftdown
		x7 = input[(row + -1) * numcols + col];			// down
		x8 = input[(row + 1) * numcols + (col + 1)];	// rightdown


		output[row * numcols + col] = (x0 * -1) + (x1 * -2) + (x3 * -1) + (x6 * 1) + (x7 * 2) + (x8 * 1);

	}
	
}

__global__ void sobelFilter(float *input, float *output, float *gradient_h_output, float *gradient_v_output, size_t width, size_t height) {
	
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;

	int numcols = WIDTH;

	float gradient,gradient_h,gradient_v;	
	float thresh = 30;	

	if (row <= height && col <= width && row > 0 && col > 0)
	{	

		gradient_h = gradient_h_output[row * numcols + col];
		gradient_v = gradient_v_output[row * numcols + col];
		gradient = sqrt((gradient_h * gradient_h) + (gradient_v * gradient_v));

		if (gradient >= thresh)
		{
			gradient = 255;
		}
		else {
			gradient = 0;
		}
		output[row * numcols + col] = gradient;
	}
}


void load_image() {
	pgmread("image512x512.pgm", (void *)image, WIDTH, HEIGHT);
	// pgmread("pgmimg.pgm", (void *)image, WIDTH, HEIGHT);
}

void save_image() {
	pgmwrite("image-output512x512.pgm", (void *)final, WIDTH, HEIGHT);
	// pgmwrite("pgmimg-output.pgm", (void *)final, WIDTH, HEIGHT);
}

void call_kernel() {
	size_t width = WIDTH, height=HEIGHT;
	int x, y;
	float *d_input, *d_output, *gradient_h_output, *gradient_v_output;

	printf("Block size: %dx%d\n", BLOCK_W, BLOCK_H);

	size_t memSize = WIDTH * HEIGHT;

	checkCudaErrors(hipMalloc(&d_input, memSize));
	checkCudaErrors(hipMalloc(&d_output, memSize));
	checkCudaErrors(hipMalloc(&gradient_h_output, memSize));
	checkCudaErrors(hipMalloc(&gradient_v_output, memSize));

	printf("Blocks per grid (width): %d |", (WIDTH / BLOCK_W));
	printf("Blocks per grid (height): %d |", (HEIGHT / BLOCK_H));

	hipStream_t streamForGraph;
  	hipGraph_t graph;
  	std::vector<hipGraphNode_t> nodeDependencies;
  	
	checkCudaErrors(hipGraphCreate(&graph, 0));
	  
	hipGraphNode_t memcpyNode;
	hipMemcpy3DParms memcpyParams = {0};

	memcpyParams.srcArray = NULL;
  	memcpyParams.srcPos = make_hipPos(0, 0, 0);
  	memcpyParams.srcPtr =
      make_hipPitchedPtr(image, memSize, 1, 1);
  	memcpyParams.dstArray = NULL;
  	memcpyParams.dstPos = make_hipPos(0, 0, 0);
  	memcpyParams.dstPtr =
      make_hipPitchedPtr(d_input, memSize, 1, 1);
  	memcpyParams.extent = make_hipExtent(memSize, 1, 1);
  	memcpyParams.kind = hipMemcpyHostToDevice;

	checkCudaErrors(
      hipGraphAddMemcpyNode(&memcpyNode, graph, NULL, 0, &memcpyParams));
	nodeDependencies.push_back(memcpyNode);

	// hipMemcpy(d_input, image, memSize, hipMemcpyHostToDevice);

	dim3 threads(BLOCK_W, BLOCK_H); // threads per block
	dim3 blocks(WIDTH / BLOCK_W, HEIGHT / BLOCK_H); // blocks per grid 

	hipGraphNode_t kernelNode;
	hipKernelNodeParams kernelNodeParams = {0};

	void* kernelArgs0[4] = {(void *)&d_input,(void *)&d_output, &width, &height};
	kernelNodeParams.func = (void *)imageBlur_horizontal;
 	kernelNodeParams.gridDim = threads;
  	kernelNodeParams.blockDim = blocks;
  	kernelNodeParams.sharedMemBytes = 0;
 	kernelNodeParams.kernelParams = (void **)kernelArgs0;
 	kernelNodeParams.extra = NULL;
  
  	//imageBlur << <blocks, threads >> > (d_input, d_output, WIDTH, HEIGHT);

	checkCudaErrors(
    hipGraphAddKernelNode(&kernelNode, graph, nodeDependencies.data(),
                             nodeDependencies.size(), &kernelNodeParams));

  	nodeDependencies.clear();
  	nodeDependencies.push_back(kernelNode);
  
  	//hipDeviceSynchronize();

	void* kernelArgs1[4] = {(void *)&d_input,(void *)&d_output, &width, &height};
	kernelNodeParams.func = (void *)imageBlur_vertical;
 	kernelNodeParams.gridDim = threads;
  	kernelNodeParams.blockDim = blocks;
  	kernelNodeParams.sharedMemBytes = 0;
 	kernelNodeParams.kernelParams = (void **)kernelArgs1;
 	kernelNodeParams.extra = NULL;

	checkCudaErrors(
    hipGraphAddKernelNode(&kernelNode, graph, nodeDependencies.data(),
                             nodeDependencies.size(), &kernelNodeParams));

  	nodeDependencies.clear();
  	nodeDependencies.push_back(kernelNode);
	
	memcpyParams.srcArray = NULL;
	memcpyParams.srcPos = make_hipPos(0, 0, 0);
	memcpyParams.srcPtr = make_hipPitchedPtr(d_input, memSize, 1, 1);
	memcpyParams.dstArray = NULL;
	memcpyParams.dstPos = make_hipPos(0, 0, 0);
	memcpyParams.dstPtr = make_hipPitchedPtr(d_output, memSize, 1, 1);
	memcpyParams.extent = make_hipExtent(memSize, 1, 1);
	memcpyParams.kind = hipMemcpyDeviceToHost;

	checkCudaErrors(
      hipGraphAddMemcpyNode(&memcpyNode, graph, nodeDependencies.data(),
                             nodeDependencies.size(), &memcpyParams));
  	nodeDependencies.clear();
  	nodeDependencies.push_back(memcpyNode);
    
  	//hipMemcpy(d_input, d_output, memSize, hipMemcpyDeviceToHost);

	void* kernelArgs2[4] = {(void *)&d_input, (void *)&gradient_h_output, &width, &height};
	kernelNodeParams.func = (void *)gradient_horizontal;
 	kernelNodeParams.gridDim = threads;
  	kernelNodeParams.blockDim = blocks;
  	kernelNodeParams.sharedMemBytes = 0;
 	kernelNodeParams.kernelParams = (void **)kernelArgs2;
 	kernelNodeParams.extra = NULL;

	checkCudaErrors(
    hipGraphAddKernelNode(&kernelNode, graph, nodeDependencies.data(),
                             nodeDependencies.size(), &kernelNodeParams));

  	nodeDependencies.clear();
  	nodeDependencies.push_back(kernelNode);

	//gradient_horizontal<< <blocks, threads>> >(d_input, gradient_h_output, WIDTH, HEIGHT);

	void* kernelArgs3[4] = {(void *)&d_input,(void *)& gradient_v_output, &width, &height};
	kernelNodeParams.func = (void *)gradient_vertical;
 	kernelNodeParams.gridDim = threads;
  	kernelNodeParams.blockDim = blocks;
  	kernelNodeParams.sharedMemBytes = 0;
 	kernelNodeParams.kernelParams = (void **)kernelArgs3;
 	kernelNodeParams.extra = NULL;

	checkCudaErrors(
    hipGraphAddKernelNode(&kernelNode, graph, nodeDependencies.data(),
                             nodeDependencies.size(), &kernelNodeParams));

  	nodeDependencies.clear();
  	nodeDependencies.push_back(kernelNode);

	//gradient_vertical<< <blocks, threads>> >(d_input, gradient_v_output, WIDTH, HEIGHT);

	void* kernelArgs4[6] = {(void *)&d_input, (void *)&d_output, (void *)&gradient_h_output, (void *)&gradient_v_output, &width, &height};
	kernelNodeParams.func = (void *)sobelFilter;
 	kernelNodeParams.gridDim = threads;
  	kernelNodeParams.blockDim = blocks;
  	kernelNodeParams.sharedMemBytes = 0;
 	kernelNodeParams.kernelParams = (void **)kernelArgs4;
 	kernelNodeParams.extra = NULL;

	checkCudaErrors(
    hipGraphAddKernelNode(&kernelNode, graph, nodeDependencies.data(),
                             nodeDependencies.size(), &kernelNodeParams));

  	nodeDependencies.clear();
  	nodeDependencies.push_back(kernelNode);

	//sobelFilter << <blocks, threads >> > (d_input, d_output, gradient_h_output, gradient_v_output, WIDTH, HEIGHT);

	//hipDeviceSynchronize();

	memcpyParams.srcArray = NULL;
	memcpyParams.srcPos = make_hipPos(0, 0, 0);
	memcpyParams.srcPtr = make_hipPitchedPtr(d_output, memSize, 1, 1);
	memcpyParams.dstArray = NULL;
	memcpyParams.dstPos = make_hipPos(0, 0, 0);
	memcpyParams.dstPtr = make_hipPitchedPtr(&final, memSize, 1, 1);
	memcpyParams.extent = make_hipExtent(memSize, 1, 1);
	memcpyParams.kind = hipMemcpyDeviceToHost;

	checkCudaErrors(
      hipGraphAddMemcpyNode(&memcpyNode, graph, nodeDependencies.data(),
                             nodeDependencies.size(), &memcpyParams));
  	nodeDependencies.clear();
  	nodeDependencies.push_back(memcpyNode);

	hipGraphExec_t graphExec;
  	checkCudaErrors(hipGraphInstantiate(&graphExec, graph, NULL, NULL, 0));
	checkCudaErrors(hipStreamSynchronize(0));

	// hipMemcpy(final, d_output, memSize, hipMemcpyDeviceToHost);

	hipError_t err = hipGetLastError();
	if (hipSuccess != err)
	{
		fprintf(stderr, "Cuda error: %s: %s.\n", "Main Loop", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	checkCudaErrors(hipGraphExecDestroy(graphExec));
  	checkCudaErrors(hipGraphDestroy(graph));
	hipFree(d_input);
	hipFree(d_output);
	hipFree(gradient_h_output);
	hipFree(gradient_v_output);
}

int main(int argc, char *argv[])
{
  hipEventCreate(&start_total);
  hipEventCreate(&stop_total);
    
  hipEventCreate(&start_sobel);
  hipEventCreate(&stop_sobel);
    
  hipEventRecord(start_total, 0);

	load_image();
   
  hipEventRecord(start_sobel, 0);

	call_kernel();
  
  hipEventRecord(stop_sobel, 0);
  hipEventSynchronize(stop_sobel);
  hipEventElapsedTime(&sobel, start_sobel, stop_sobel);

	save_image();
   
  hipEventRecord(stop_total, 0);
  hipEventSynchronize(stop_total);
  hipEventElapsedTime(&total, start_total, stop_total);
    
  printf("Total Parallel Time:  %f s |", sobel/1000);
  printf("Total Serial Time:  %f s |", (total-sobel)/1000);
  printf("Total Time:  %f s |", total/1000);
  
    
	hipDeviceReset();
	
	return 0;
}

