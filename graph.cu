#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#ifdef __INTELLISENSE__
void __syncthreads();
#endif

#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime_api.h>
#include <pgmio.h>
#include <vector>

// image dimensions WIDTH & HEIGHT
#define WIDTH 256
#define HEIGHT 256

// Block width WIDTH & HEIGHT
#define BLOCK_W 32
#define BLOCK_H 32

// prototype declarations

void load_image(float *image);
void call_kernel(float *image, float *final, int devID);
void save_image(float *final);
void prepareAllocParams(hipMemAllocNodeParams *allocParams, size_t bytes,int device);

#define MAXLINE 128

float total, sobel;
hipEvent_t start_total, stop_total;
hipEvent_t start_sobel, stop_sobel;


__global__ void imageBlur_horizontal(float *input, float *output, size_t width, size_t height) {

	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;

	int numcols = WIDTH;

	float blur;

	if (row <= height && col <= width && row > 0 && col > 0)
	{
		// weights
		int	x3, x4, x5;

		// blur
		// 0.2 0.2 0.2

		x3 = input[row * numcols + (col - 1)];			// left
		x4 = input[row * numcols + col];				// center
		x5 = input[row * numcols + (col + 1)];			// right

		blur =  (x3 * 0.2) + (x4 * 0.2) + (x5 * 0.2);

		output[row * numcols + col] = blur;
	}
}

__global__ void imageBlur_vertical(float *input, float *output, size_t width, size_t height) {
	
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;

	int numcols = WIDTH;

	float blur;

	if (row <= height && col <= width && row > 0 && col > 0)
	{
		// weights
		int		x1,x7;

		// blur
		// 0.0 0.2 0.0
		// 0.2 0.2 0.2
		// 0.0 0.2 0.0

		x1 = input[(row + 1) * numcols + col];			// up
		x7 = input[(row + -1) * numcols + col];			// down

		blur = (x1 * 0.2) + (x7 * 0.2);

		output[row * numcols + col] = blur;
	}
}

__global__ void gradient_horizontal(float *input, float *output, size_t width, size_t height) {

	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;

	int numcols = WIDTH;


	// horizontal
	// -1  0  1
	// -2  0  2
	// -1  0  1

	int x0, x2, 
		x3, x5, 
		x6, x8;

	if (row <= height && col <= width && row > 0 && col > 0)
	{
	
	x0 = input[(row - 1) * numcols + (col - 1)];	// leftup
	x2 = input[(row - 1) * numcols + (col + 1)];	// rightup
	x3 = input[row * numcols + (col - 1)];			// left
	x5 = input[row * numcols + (col + 1)];			// right
	x6 = input[(row + 1) * numcols + (col - 1)];	// leftdown
	x8 = input[(row + 1) * numcols + (col + 1)];	// rightdown


	output[row * numcols + col] = (x0 * -1) + (x2 * 1) + (x3 * -2) + (x5 * 2) + (x6 * -1) + (x8 * 1);

	}

	return;
}


__global__ void gradient_vertical(float *input, float *output, size_t width, size_t height) {

	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;

	int numcols = WIDTH;

	// vertical 
	// -1 -2 -1
	//  0  0  0
	//  1  2  1

	int x0, x1, 
		x3, x6, 
		x7, x8;

	if (row <= height && col <= width && row > 0 && col > 0)
	{
		x0 = input[(row - 1) * numcols + (col - 1)];	// leftup
		x1 = input[(row + 1) * numcols + col];			// up
		x3 = input[row * numcols + (col - 1)];			// left
		x6 = input[(row + 1) * numcols + (col - 1)];	// leftdown
		x7 = input[(row + -1) * numcols + col];			// down
		x8 = input[(row + 1) * numcols + (col + 1)];	// rightdown


		output[row * numcols + col] = (x0 * -1) + (x1 * -2) + (x3 * -1) + (x6 * 1) + (x7 * 2) + (x8 * 1);

	}
	
}

__global__ void sobelFilter(float *input, float *output, float *gradient_h_output, float *gradient_v_output, size_t width, size_t height) {
	
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;

	int numcols = WIDTH;

	float gradient,gradient_h,gradient_v;	
	float thresh = 30;	

	if (row <= height && col <= width && row > 0 && col > 0)
	{	

		gradient_h = gradient_h_output[row * numcols + col];
		gradient_v = gradient_v_output[row * numcols + col];
		gradient = sqrt((gradient_h * gradient_h) + (gradient_v * gradient_v));

		if (gradient >= thresh)
		{
			gradient = 255;
		}
		else {
			gradient = 0;
		}
		output[row * numcols + col] = gradient;
	}
}


void load_image(float *image) {
	// pgmread("image100000x100000.pgm", (void *)image, WIDTH, HEIGHT);
	// pgmread("image16384x16384.pgm", (void *)image, WIDTH, HEIGHT);
	// pgmread("image10000x10000.pgm", (void *)image, WIDTH, HEIGHT);
	// pgmread("image4096x4096.pgm", (void *)image, WIDTH, HEIGHT);
	// pgmread("image2048x2048.pgm", (void *)image, WIDTH, HEIGHT);
	// pgmread("image1024x1024.pgm", (void *)image, WIDTH, HEIGHT);
	pgmread("image512x512.pgm", (void *)image, WIDTH, HEIGHT);
	// pgmread("pgmimg.pgm", (void *)image, WIDTH, HEIGHT);
}

void save_image(float *final) {
	// pgmwrite("image-outputl100000x100000.pgm", (void *)final, WIDTH, HEIGHT);
	// pgmwrite("image-outputl16384x16384.pgm", (void *)final, WIDTH, HEIGHT);
	// pgmwrite("image-outputl10000x1000.pgm", (void *)final, WIDTH, HEIGHT);
	// pgmwrite("image-outputl4096x4096.pgm", (void *)final, WIDTH, HEIGHT);
	// pgmwrite("image-outputl2048x2048.pgm", (void *)final, WIDTH, HEIGHT);
	// pgmwrite("image-outputl1024x1024.pgm", (void *)final, WIDTH, HEIGHT);
	pgmwrite("image-output_g_512x512.pgm", (void *)final, WIDTH, HEIGHT);
	// pgmwrite("pgmimg-output.pgm", (void *)final, WIDTH, HEIGHT);
}

void prepareAllocParams(hipMemAllocNodeParams *allocParams, size_t bytes,
                        int device) {
  memset(allocParams, 0, sizeof(*allocParams));

  allocParams->bytesize = bytes;
  allocParams->poolProps.allocType = hipMemAllocationTypePinned;
  allocParams->poolProps.location.id = device;
  allocParams->poolProps.location.type = hipMemLocationTypeDevice;
}

void call_kernel(float *image, float *final, int devID) {
	size_t width = WIDTH, height = HEIGHT;
	int x, y;
	float *d_input, *d_output, *gradient_h_output, *gradient_v_output;

	printf("Block size: %dx%d\n", BLOCK_W, BLOCK_H);

	size_t memSize = WIDTH * HEIGHT;

	checkCudaErrors(hipMalloc(&d_input, memSize));
	checkCudaErrors(hipMalloc(&d_output, memSize));
	checkCudaErrors(hipMalloc(&gradient_h_output, memSize));
	checkCudaErrors(hipMalloc(&gradient_v_output, memSize));

	printf("Blocks per grid (width): %d |", (WIDTH / BLOCK_W));
	printf("Blocks per grid (height): %d \n", (HEIGHT / BLOCK_H));

  	hipGraph_t graph;
  	std::vector<hipGraphNode_t> nodeDependencies;

	checkCudaErrors(hipGraphCreate(&graph, 0));

	// hipGraphNode_t memAllocNode,memFreeNode;
	// hipMemAllocNodeParams memAllocParams;

	// prepareAllocParams(&memAllocParams, memSize, devID);
	// checkCudaErrors(
	// 	hipGraphAddMemAllocNode(&memAllocNode, graph, NULL, 0, &memAllocParams));
	
	// d_input = (float *)memAllocParams.dptr;

	// checkCudaErrors(
    //   hipGraphAddMemFreeNode(&memFreeNode, graph, &memAllocNode, 1, (void *)d_input));
	
	hipGraphNode_t memcpyNode;
	hipMemcpy3DParms memcpyParams = {0};

	memcpyParams.srcArray = NULL;
  	memcpyParams.srcPos = make_hipPos(0, 0, 0);
  	memcpyParams.srcPtr =
      make_hipPitchedPtr(&image, memSize, 1, 1);
  	memcpyParams.dstArray = NULL;
  	memcpyParams.dstPos = make_hipPos(0, 0, 0);
  	memcpyParams.dstPtr =
      make_hipPitchedPtr(d_input, memSize, 1, 1);
  	memcpyParams.extent = make_hipExtent(memSize, 1, 1);
  	memcpyParams.kind = hipMemcpyHostToDevice;

	checkCudaErrors(
      hipGraphAddMemcpyNode(&memcpyNode, graph, NULL, 0, &memcpyParams));
	nodeDependencies.push_back(memcpyNode);

	// hipMemcpy(d_input, image, memSize, hipMemcpyHostToDevice);

	dim3 threads(BLOCK_W, BLOCK_H); // threads per block
	dim3 blocks(WIDTH / BLOCK_W, HEIGHT / BLOCK_H); // blocks per grid 

	hipGraphNode_t kernelNode;
	hipKernelNodeParams kernelNodeParams = {0};

	void* kernelArgs0[4] = {(void *)&d_input,(void *)&d_output, &width, &height};
	kernelNodeParams.func = (void *)imageBlur_horizontal;
 	kernelNodeParams.gridDim = blocks;
  	kernelNodeParams.blockDim = threads;
  	kernelNodeParams.sharedMemBytes = 0;
 	kernelNodeParams.kernelParams = (void **)kernelArgs0;
 	kernelNodeParams.extra = NULL;
  
  	//imageBlur << <blocks, threads >> > (d_input, d_output, WIDTH, HEIGHT);

	checkCudaErrors(
    hipGraphAddKernelNode(&kernelNode, graph, nodeDependencies.data(),
                             nodeDependencies.size(), &kernelNodeParams));

  	nodeDependencies.clear();
  	nodeDependencies.push_back(kernelNode);
  
  	//hipDeviceSynchronize();

	hipKernelNodeParams kernelNodeParams1 = {0};
	void* kernelArgs1[4] = {(void *)&d_input,(void *)&d_output, &width, &height};
	kernelNodeParams1.func = (void *)imageBlur_vertical;
 	kernelNodeParams1.gridDim = blocks;
  	kernelNodeParams1.blockDim = threads;
  	kernelNodeParams1.sharedMemBytes = 0;
 	kernelNodeParams1.kernelParams = (void **)kernelArgs1;
 	kernelNodeParams1.extra = NULL;

	checkCudaErrors(
    hipGraphAddKernelNode(&kernelNode, graph, nodeDependencies.data(),
                             nodeDependencies.size(), &kernelNodeParams1));

  	nodeDependencies.clear();
  	nodeDependencies.push_back(kernelNode);
	
	hipMemcpy3DParms memcpyParams1 = {0};
	memcpyParams1.srcArray = NULL;
	memcpyParams1.srcPos = make_hipPos(0, 0, 0);
	memcpyParams1.srcPtr = make_hipPitchedPtr(d_input, memSize, 1, 1);
	memcpyParams1.dstArray = NULL;
	memcpyParams1.dstPos = make_hipPos(0, 0, 0);
	memcpyParams1.dstPtr = make_hipPitchedPtr(d_output, memSize, 1, 1);
	memcpyParams1.extent = make_hipExtent(memSize, 1, 1);
	memcpyParams1.kind = hipMemcpyDeviceToHost;

	checkCudaErrors(
      hipGraphAddMemcpyNode(&memcpyNode, graph, nodeDependencies.data(),
                             nodeDependencies.size(), &memcpyParams1));
  	nodeDependencies.clear();
  	nodeDependencies.push_back(memcpyNode);
    
  	//hipMemcpy(d_input, d_output, memSize, hipMemcpyDeviceToHost);

	hipKernelNodeParams kernelNodeParams2 = {0};
	void* kernelArgs2[4] = {(void *)&d_input, (void *)&gradient_h_output, &width, &height};
	kernelNodeParams2.func = (void *)gradient_horizontal;
 	kernelNodeParams2.gridDim = blocks;
  	kernelNodeParams2.blockDim = threads;
  	kernelNodeParams2.sharedMemBytes = 0;
 	kernelNodeParams2.kernelParams = (void **)kernelArgs2;
 	kernelNodeParams2.extra = NULL;

	checkCudaErrors(
    hipGraphAddKernelNode(&kernelNode, graph, nodeDependencies.data(),
                             nodeDependencies.size(), &kernelNodeParams2));

  	nodeDependencies.clear();
  	nodeDependencies.push_back(kernelNode);

	//gradient_horizontal<< <blocks, threads>> >(d_input, gradient_h_output, WIDTH, HEIGHT);
	
	hipKernelNodeParams kernelNodeParams3 = {0};
	void* kernelArgs3[4] = {(void *)&d_input,(void *)& gradient_v_output, &width, &height};
	kernelNodeParams3.func = (void *)gradient_vertical;
 	kernelNodeParams3.gridDim = blocks;
  	kernelNodeParams3.blockDim = threads;
  	kernelNodeParams3.sharedMemBytes = 0;
 	kernelNodeParams3.kernelParams = (void **)kernelArgs3;
 	kernelNodeParams3.extra = NULL;

	checkCudaErrors(
    hipGraphAddKernelNode(&kernelNode, graph, nodeDependencies.data(),
                             nodeDependencies.size(), &kernelNodeParams3));

  	nodeDependencies.clear();
  	nodeDependencies.push_back(kernelNode);

	//gradient_vertical<< <blocks, threads>> >(d_input, gradient_v_output, WIDTH, HEIGHT);
	hipKernelNodeParams kernelNodeParams4 = {0};
	void* kernelArgs4[6] = {(void *)&d_input, (void *)&d_output, (void *)&gradient_h_output, (void *)&gradient_v_output, &width, &height};
	kernelNodeParams4.func = (void *)sobelFilter;
 	kernelNodeParams4.gridDim = blocks;
  	kernelNodeParams4.blockDim = threads;
  	kernelNodeParams4.sharedMemBytes = 0;
 	kernelNodeParams4.kernelParams = (void **)kernelArgs4;
 	kernelNodeParams4.extra = NULL;

	checkCudaErrors(
    hipGraphAddKernelNode(&kernelNode, graph, nodeDependencies.data(),
                             nodeDependencies.size(), &kernelNodeParams4));

  	nodeDependencies.clear();
  	nodeDependencies.push_back(kernelNode);

	//sobelFilter << <blocks, threads >> > (d_input, d_output, gradient_h_output, gradient_v_output, WIDTH, HEIGHT);

	//hipDeviceSynchronize();
	hipMemcpy3DParms memcpyParams2 = {0};
	memcpyParams2.srcArray = NULL;
	memcpyParams2.srcPos = make_hipPos(0, 0, 0);
	memcpyParams2.srcPtr = make_hipPitchedPtr(d_output, memSize, 1, 1);
	memcpyParams2.dstArray = NULL;
	memcpyParams2.dstPos = make_hipPos(0, 0, 0);
	memcpyParams2.dstPtr = make_hipPitchedPtr(&final, memSize, 1, 1);
	memcpyParams2.extent = make_hipExtent(memSize, 1, 1);
	memcpyParams2.kind = hipMemcpyDeviceToHost;

	checkCudaErrors(
      hipGraphAddMemcpyNode(&memcpyNode, graph, nodeDependencies.data(),
                             nodeDependencies.size(), &memcpyParams2));
  	nodeDependencies.clear();
  	nodeDependencies.push_back(memcpyNode);


	checkCudaErrors(hipGraphDebugDotPrint(graph, "mainGraph.dot", 0));

	hipGraphExec_t graphExec;
  	checkCudaErrors(hipGraphInstantiate(&graphExec, graph, NULL, NULL, 0));
	checkCudaErrors(hipGraphLaunch(graphExec, 0));
	checkCudaErrors(hipStreamSynchronize(0));

	// hipMemcpy(final, d_output, memSize, hipMemcpyDeviceToHost);

	hipError_t err = hipGetLastError();
	if (hipSuccess != err)
	{
		fprintf(stderr, "Cuda error: %s: %s.\n", "Main Loop", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	checkCudaErrors(hipGraphExecDestroy(graphExec));
  	checkCudaErrors(hipGraphDestroy(graph));
	hipFree(d_input);
	hipFree(d_output);
	hipFree(gradient_h_output);
	hipFree(gradient_v_output);
}

int main(int argc, char *argv[])
{
 // This will pick the best possible CUDA capable device
 int devID = findCudaDevice(argc, (const char **)argv);
 float *image = NULL, *final = NULL;
 size_t memSize = WIDTH * HEIGHT * sizeof(float);
 checkCudaErrors((hipHostMalloc(&image, memSize)));
 checkCudaErrors((hipHostMalloc(&final, memSize)));

  hipEventCreate(&start_total);
  hipEventCreate(&stop_total);
    
  hipEventCreate(&start_sobel);
  hipEventCreate(&stop_sobel);
    
  hipEventRecord(start_total, 0);

	load_image(image);
   
  hipEventRecord(start_sobel, 0);

	call_kernel(image,final,devID);
  
  hipEventRecord(stop_sobel, 0);
  hipEventSynchronize(stop_sobel);
  hipEventElapsedTime(&sobel, start_sobel, stop_sobel);

	save_image(final);
   
  hipEventRecord(stop_total, 0);
  hipEventSynchronize(stop_total);
  hipEventElapsedTime(&total, start_total, stop_total);
    
  printf("Total Parallel Time:  %f s \n", sobel/1000);
  printf("Total Serial Time:  %f s \n", (total-sobel)/1000);
  printf("Total Time:  %f s \n", total/1000);
  
    
	hipDeviceReset();
	
	return 0;
}

