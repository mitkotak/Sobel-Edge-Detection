#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#ifdef __INTELLISENSE__
void __syncthreads();
#endif

#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime_api.h>
#include <pgmio.h>
#include <vector>

// Block width WIDTH & HEIGHT
#define BLOCK_W 10
#define BLOCK_H 10

// prototype declarations

#define MAXLINE 128

float total, sobel;
hipEvent_t start_total, stop_total;
hipEvent_t start_sobel, stop_sobel;

__global__ void warm_up_gpu(){
  unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
  float ia, ib;
  ia = ib = 0.0f;
  ib += ia + tid; 
}


__global__ void imageBlur_horizontal(float *input, float *output, size_t width, size_t height) {

	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;

	int numcols = width;

	float blur;

	if (row <= height && col <= width && row > 0 && col > 0)
	{
		// weights
		int	x3, x4, x5;

		// blur
		// 0.2 0.2 0.2

		x3 = input[row * numcols + (col - 1)];			// left
		x4 = input[row * numcols + col];				// center
		x5 = input[row * numcols + (col + 1)];			// right

		blur =  (x3 * 0.2) + (x4 * 0.2) + (x5 * 0.2);

		output[row * numcols + col] = blur;
	}
}

__global__ void imageBlur_vertical(float *input, float *output, size_t width, size_t height) {
	
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;

	int numcols = width;

	float blur;

	if (row <= height && col <= width && row > 0 && col > 0)
	{
		// weights
		int		x1,x7;

		// blur
		// 0.0 0.2 0.0
		// 0.2 0.2 0.2
		// 0.0 0.2 0.0

		x1 = input[(row + 1) * numcols + col];			// up
		x7 = input[(row + -1) * numcols + col];			// down

		blur = (x1 * 0.2) + (x7 * 0.2);

		output[row * numcols + col] = blur;
	}
}

__global__ void gradient_horizontal(float *input, float *output, size_t width, size_t height) {

	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;

	int numcols = width;


	// horizontal
	// -1  0  1
	// -2  0  2
	// -1  0  1

	int x0, x2, 
		x3, x5, 
		x6, x8;

	if (row <= height && col <= width && row > 0 && col > 0)
	{
	
	x0 = input[(row - 1) * numcols + (col - 1)];	// leftup
	x2 = input[(row - 1) * numcols + (col + 1)];	// rightup
	x3 = input[row * numcols + (col - 1)];			// left
	x5 = input[row * numcols + (col + 1)];			// right
	x6 = input[(row + 1) * numcols + (col - 1)];	// leftdown
	x8 = input[(row + 1) * numcols + (col + 1)];	// rightdown


	output[row * numcols + col] = (x0 * -1) + (x2 * 1) + (x3 * -2) + (x5 * 2) + (x6 * -1) + (x8 * 1);

	}

	return;
}


__global__ void gradient_vertical(float *input, float *output, size_t width, size_t height) {

	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;

	int numcols = width;

	// vertical 
	// -1 -2 -1
	//  0  0  0
	//  1  2  1

	int x0, x1, 
		x3, x6, 
		x7, x8;

	if (row <= height && col <= width && row > 0 && col > 0)
	{
		x0 = input[(row - 1) * numcols + (col - 1)];	// leftup
		x1 = input[(row + 1) * numcols + col];			// up
		x3 = input[row * numcols + (col - 1)];			// left
		x6 = input[(row + 1) * numcols + (col - 1)];	// leftdown
		x7 = input[(row + -1) * numcols + col];			// down
		x8 = input[(row + 1) * numcols + (col + 1)];	// rightdown


		output[row * numcols + col] = (x0 * -1) + (x1 * -2) + (x3 * -1) + (x6 * 1) + (x7 * 2) + (x8 * 1);

	}
	
}

__global__ void sobelFilter(float *input, float *output, float *gradient_h_output, float *gradient_v_output, size_t width, size_t height) {
	
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;

	int numcols = width;

	float gradient,gradient_h,gradient_v;	
	float thresh = 30;	

	if (row <= height && col <= width && row > 0 && col > 0)
	{	

		gradient_h = gradient_h_output[row * numcols + col];
		gradient_v = gradient_v_output[row * numcols + col];
		gradient = sqrt((gradient_h * gradient_h) + (gradient_v * gradient_v));

		if (gradient >= thresh)
		{
			gradient = 255;
		}
		else {
			gradient = 0;
		}
		output[row * numcols + col] = gradient;
	}
}

int main(int argc, char *argv[])
{
	size_t width = 600, height = 600;
	int devID = findCudaDevice(argc, (const char **)argv);
	float *image = NULL, *final = NULL;
	float *image2 = NULL, *final2 = NULL;
	float *image3 = NULL, *final3 = NULL;
	float *image4 = NULL, *final4 = NULL;
	float *image5 = NULL, *final5 = NULL;
	float *image6 = NULL, *final6 = NULL;
	float *image7 = NULL, *final7 = NULL;
	float *image8 = NULL, *final8 = NULL;

	size_t memSize = width * height * sizeof(float);
	checkCudaErrors((hipHostMalloc(&image, memSize)));
	checkCudaErrors((hipHostMalloc(&image2, memSize)));
	checkCudaErrors((hipHostMalloc(&final, memSize)));
	checkCudaErrors((hipHostMalloc(&final2, memSize)));
	checkCudaErrors((hipHostMalloc(&image3, memSize)));
	checkCudaErrors((hipHostMalloc(&image4, memSize)));
	checkCudaErrors((hipHostMalloc(&final3, memSize)));
	checkCudaErrors((hipHostMalloc(&final4, memSize)));
	checkCudaErrors((hipHostMalloc(&image5, memSize)));
	checkCudaErrors((hipHostMalloc(&image6, memSize)));
	checkCudaErrors((hipHostMalloc(&final5, memSize)));
	checkCudaErrors((hipHostMalloc(&final6, memSize)));
	checkCudaErrors((hipHostMalloc(&image7, memSize)));
	checkCudaErrors((hipHostMalloc(&image8, memSize)));
	checkCudaErrors((hipHostMalloc(&final7, memSize)));
	checkCudaErrors((hipHostMalloc(&final8, memSize)));
	// read image 
	pgmread("../images/test_images/apollonian_gasket.ascii.pgm", (void *)image, width, height);
	pgmread("../images/test_images/apollonian_gasket.ascii.pgm", (void *)image2, width, height);
	pgmread("../images/test_images/apollonian_gasket.ascii.pgm", (void *)image3, width, height);
	pgmread("../images/test_images/apollonian_gasket.ascii.pgm", (void *)image4, width, height);
	pgmread("../images/test_images/apollonian_gasket.ascii.pgm", (void *)image5, width, height);
	pgmread("../images/test_images/apollonian_gasket.ascii.pgm", (void *)image6, width, height);
	pgmread("../images/test_images/apollonian_gasket.ascii.pgm", (void *)image7, width, height);
	pgmread("../images/test_images/apollonian_gasket.ascii.pgm", (void *)image8, width, height);
	
	hipEventCreate(&start_total);
	hipEventCreate(&stop_total);
	hipEventRecord(start_total, 0);

	int x, y;
	float *d_input, *d_output, *gradient_h_output, *gradient_v_output;
	float *d_input2, *d_output2, *gradient_h_output2, *gradient_v_output2;
	float *d_input3, *d_output3, *gradient_h_output3, *gradient_v_output3;
	float *d_input4, *d_output4, *gradient_h_output4, *gradient_v_output4;
	float *d_input5, *d_output5, *gradient_h_output5, *gradient_v_output5;
	float *d_input6, *d_output6, *gradient_h_output6, *gradient_v_output6;
	float *d_input7, *d_output7, *gradient_h_output7, *gradient_v_output7;
	float *d_input8, *d_output8, *gradient_h_output8, *gradient_v_output8;

	printf("Block size: %dx%d\n", BLOCK_W, BLOCK_H);

	checkCudaErrors(hipMalloc(&d_input, memSize));
	checkCudaErrors(hipMalloc(&d_output, memSize));
	checkCudaErrors(hipMalloc(&gradient_h_output, memSize));
	checkCudaErrors(hipMalloc(&gradient_v_output, memSize));

	checkCudaErrors(hipMalloc(&d_input2, memSize));
	checkCudaErrors(hipMalloc(&d_output2, memSize));
	checkCudaErrors(hipMalloc(&gradient_h_output2, memSize));
	checkCudaErrors(hipMalloc(&gradient_v_output2, memSize));

	checkCudaErrors(hipMalloc(&d_input3, memSize));
	checkCudaErrors(hipMalloc(&d_output3, memSize));
	checkCudaErrors(hipMalloc(&gradient_h_output3, memSize));
	checkCudaErrors(hipMalloc(&gradient_v_output3, memSize));

	checkCudaErrors(hipMalloc(&d_input4, memSize));
	checkCudaErrors(hipMalloc(&d_output4, memSize));
	checkCudaErrors(hipMalloc(&gradient_h_output4, memSize));
	checkCudaErrors(hipMalloc(&gradient_v_output4, memSize));

	checkCudaErrors(hipMalloc(&d_input5, memSize));
	checkCudaErrors(hipMalloc(&d_output5, memSize));
	checkCudaErrors(hipMalloc(&gradient_h_output5, memSize));
	checkCudaErrors(hipMalloc(&gradient_v_output5, memSize));

	checkCudaErrors(hipMalloc(&d_input6, memSize));
	checkCudaErrors(hipMalloc(&d_output6, memSize));
	checkCudaErrors(hipMalloc(&gradient_h_output6, memSize));
	checkCudaErrors(hipMalloc(&gradient_v_output6, memSize));

	checkCudaErrors(hipMalloc(&d_input7, memSize));
	checkCudaErrors(hipMalloc(&d_output7, memSize));
	checkCudaErrors(hipMalloc(&gradient_h_output7, memSize));
	checkCudaErrors(hipMalloc(&gradient_v_output7, memSize));

	checkCudaErrors(hipMalloc(&d_input8, memSize));
	checkCudaErrors(hipMalloc(&d_output8, memSize));
	checkCudaErrors(hipMalloc(&gradient_h_output8, memSize));
	checkCudaErrors(hipMalloc(&gradient_v_output8, memSize));


	printf("Blocks per grid (width): %d |", (width / BLOCK_W));
	printf("Blocks per grid (height): %d \n", (height / BLOCK_H));

  	hipGraph_t graph;
  	std::vector<hipGraphNode_t> nodeDependencies;
	std::vector<hipGraphNode_t> nodeDependencies2;
	std::vector<hipGraphNode_t> nodeDependencies3;
	std::vector<hipGraphNode_t> nodeDependencies4;
	std::vector<hipGraphNode_t> nodeDependencies5;
	std::vector<hipGraphNode_t> nodeDependencies6;
	std::vector<hipGraphNode_t> nodeDependencies7;
	std::vector<hipGraphNode_t> nodeDependencies8;

	checkCudaErrors(hipGraphCreate(&graph, 0));

	// Copy data from host to device
	hipGraphNode_t memcpyNode;
	hipMemcpy3DParms memcpyParams = {0};

	memcpyParams.srcArray = NULL;
  	memcpyParams.srcPos = make_hipPos(0, 0, 0);
  	memcpyParams.srcPtr =
      make_hipPitchedPtr(image, memSize, 1, 1);
  	memcpyParams.dstArray = NULL;
  	memcpyParams.dstPos = make_hipPos(0, 0, 0);
  	memcpyParams.dstPtr =
      make_hipPitchedPtr(d_input, memSize, 1, 1);
  	memcpyParams.extent = make_hipExtent(memSize, 1, 1);
  	memcpyParams.kind = hipMemcpyHostToDevice;

	checkCudaErrors(
      hipGraphAddMemcpyNode(&memcpyNode, graph, NULL, 0, &memcpyParams));
	nodeDependencies.push_back(memcpyNode);

	hipMemcpy3DParms memcpyParams01 = {0};

	memcpyParams01.srcArray = NULL;
  	memcpyParams01.srcPos = make_hipPos(0, 0, 0);
  	memcpyParams01.srcPtr =
      make_hipPitchedPtr(image2, memSize, 1, 1);
  	memcpyParams01.dstArray = NULL;
  	memcpyParams01.dstPos = make_hipPos(0, 0, 0);
  	memcpyParams01.dstPtr =
      make_hipPitchedPtr(d_input2, memSize, 1, 1);
  	memcpyParams01.extent = make_hipExtent(memSize, 1, 1);
  	memcpyParams01.kind = hipMemcpyHostToDevice;

	hipGraphNode_t memcpyNode2;
	checkCudaErrors(
      hipGraphAddMemcpyNode(&memcpyNode2, graph, NULL, 0, &memcpyParams01));
	nodeDependencies2.push_back(memcpyNode2);

	
	hipMemcpy3DParms memcpyParams02 = {0};

	memcpyParams02.srcArray = NULL;
  	memcpyParams02.srcPos = make_hipPos(0, 0, 0);
  	memcpyParams02.srcPtr =
      make_hipPitchedPtr(image3, memSize, 1, 1);
  	memcpyParams02.dstArray = NULL;
  	memcpyParams02.dstPos = make_hipPos(0, 0, 0);
  	memcpyParams02.dstPtr =
      make_hipPitchedPtr(d_input3, memSize, 1, 1);
  	memcpyParams02.extent = make_hipExtent(memSize, 1, 1);
  	memcpyParams02.kind = hipMemcpyHostToDevice;

	hipGraphNode_t memcpyNode3;
	checkCudaErrors(
      hipGraphAddMemcpyNode(&memcpyNode3, graph, NULL, 0, &memcpyParams02));
	nodeDependencies3.push_back(memcpyNode3);

	hipMemcpy3DParms memcpyParams03 = {0};
	memcpyParams03.srcArray = NULL;
  	memcpyParams03.srcPos = make_hipPos(0, 0, 0);
  	memcpyParams03.srcPtr =
      make_hipPitchedPtr(image4, memSize, 1, 1);
  	memcpyParams03.dstArray = NULL;
  	memcpyParams03.dstPos = make_hipPos(0, 0, 0);
  	memcpyParams03.dstPtr =
      make_hipPitchedPtr(d_input4, memSize, 1, 1);
  	memcpyParams03.extent = make_hipExtent(memSize, 1, 1);
  	memcpyParams03.kind = hipMemcpyHostToDevice;

	hipGraphNode_t memcpyNode4;
	checkCudaErrors(
      hipGraphAddMemcpyNode(&memcpyNode4, graph, NULL, 0, &memcpyParams03));
	nodeDependencies4.push_back(memcpyNode4);

	hipGraphNode_t memcpyNode5;
	hipMemcpy3DParms memcpyParams04 = {0};

	memcpyParams04.srcArray = NULL;
  	memcpyParams04.srcPos = make_hipPos(0, 0, 0);
  	memcpyParams04.srcPtr =
      make_hipPitchedPtr(image5, memSize, 1, 1);
  	memcpyParams04.dstArray = NULL;
  	memcpyParams04.dstPos = make_hipPos(0, 0, 0);
  	memcpyParams04.dstPtr =
      make_hipPitchedPtr(d_input5, memSize, 1, 1);
  	memcpyParams04.extent = make_hipExtent(memSize, 1, 1);
  	memcpyParams04.kind = hipMemcpyHostToDevice;

	checkCudaErrors(
      hipGraphAddMemcpyNode(&memcpyNode5, graph, NULL, 0, &memcpyParams04));
	nodeDependencies5.push_back(memcpyNode5);

	hipMemcpy3DParms memcpyParams05 = {0};

	memcpyParams05.srcArray = NULL;
  	memcpyParams05.srcPos = make_hipPos(0, 0, 0);
  	memcpyParams05.srcPtr =
      make_hipPitchedPtr(image6, memSize, 1, 1);
  	memcpyParams05.dstArray = NULL;
  	memcpyParams05.dstPos = make_hipPos(0, 0, 0);
  	memcpyParams05.dstPtr =
      make_hipPitchedPtr(d_input6, memSize, 1, 1);
  	memcpyParams05.extent = make_hipExtent(memSize, 1, 1);
  	memcpyParams05.kind = hipMemcpyHostToDevice;

	hipGraphNode_t memcpyNode6;
	checkCudaErrors(
      hipGraphAddMemcpyNode(&memcpyNode6, graph, NULL, 0, &memcpyParams05));
	nodeDependencies6.push_back(memcpyNode6);

	
	hipMemcpy3DParms memcpyParams06 = {0};

	memcpyParams06.srcArray = NULL;
  	memcpyParams06.srcPos = make_hipPos(0, 0, 0);
  	memcpyParams06.srcPtr =
      make_hipPitchedPtr(image7, memSize, 1, 1);
  	memcpyParams06.dstArray = NULL;
  	memcpyParams06.dstPos = make_hipPos(0, 0, 0);
  	memcpyParams06.dstPtr =
      make_hipPitchedPtr(d_input7, memSize, 1, 1);
  	memcpyParams06.extent = make_hipExtent(memSize, 1, 1);
  	memcpyParams06.kind = hipMemcpyHostToDevice;

	hipGraphNode_t memcpyNode7;
	checkCudaErrors(
      hipGraphAddMemcpyNode(&memcpyNode7, graph, NULL, 0, &memcpyParams06));
	nodeDependencies7.push_back(memcpyNode7);

	hipMemcpy3DParms memcpyParams07 = {0};
	memcpyParams07.srcArray = NULL;
  	memcpyParams07.srcPos = make_hipPos(0, 0, 0);
  	memcpyParams07.srcPtr =
      make_hipPitchedPtr(image8, memSize, 1, 1);
  	memcpyParams07.dstArray = NULL;
  	memcpyParams07.dstPos = make_hipPos(0, 0, 0);
  	memcpyParams07.dstPtr =
      make_hipPitchedPtr(d_input8, memSize, 1, 1);
  	memcpyParams07.extent = make_hipExtent(memSize, 1, 1);
  	memcpyParams07.kind = hipMemcpyHostToDevice;

	hipGraphNode_t memcpyNode8;
	checkCudaErrors(
      hipGraphAddMemcpyNode(&memcpyNode8, graph, NULL, 0, &memcpyParams07));
	nodeDependencies8.push_back(memcpyNode8);

	dim3 threads(BLOCK_W, BLOCK_H); // threads per block
	dim3 blocks(width / BLOCK_W, height / BLOCK_H); // blocks per grid 

	// imageBlur_horizontal kernel
	hipGraphNode_t kernelNode;
	hipKernelNodeParams kernelNodeParams = {0};

	void* kernelArgs0[4] = {(void *)&d_input,(void *)&d_output, &width, &height};
	kernelNodeParams.func = (void *)imageBlur_horizontal;
 	kernelNodeParams.gridDim = blocks;
  	kernelNodeParams.blockDim = threads;
  	kernelNodeParams.sharedMemBytes = 0;
 	kernelNodeParams.kernelParams = (void **)kernelArgs0;
 	kernelNodeParams.extra = NULL;


	checkCudaErrors(
    hipGraphAddKernelNode(&kernelNode, graph, &memcpyNode,
                             1, &kernelNodeParams));

  	nodeDependencies.clear();
  	nodeDependencies.push_back(kernelNode);
  
	hipGraphNode_t kernelNode2;
	hipKernelNodeParams kernelNodeParams01 = {0};

	void* kernelArgs01[4] = {(void *)&d_input2,(void *)&d_output2, &width, &height};
	kernelNodeParams01.func = (void *)imageBlur_horizontal;
 	kernelNodeParams01.gridDim = blocks;
  	kernelNodeParams01.blockDim = threads;
  	kernelNodeParams01.sharedMemBytes = 0;
 	kernelNodeParams01.kernelParams = (void **)kernelArgs01;
 	kernelNodeParams01.extra = NULL;

	checkCudaErrors(
    hipGraphAddKernelNode(&kernelNode2, graph, &memcpyNode2,
                             1, &kernelNodeParams01));
  	nodeDependencies2.clear();
  	nodeDependencies2.push_back(kernelNode2);

	hipGraphNode_t kernelNode3;
	hipKernelNodeParams kernelNodeParams02 = {0};

	void* kernelArgs02[4] = {(void *)&d_input3,(void *)&d_output3, &width, &height};
	kernelNodeParams02.func = (void *)imageBlur_horizontal;
 	kernelNodeParams02.gridDim = blocks;
  	kernelNodeParams02.blockDim = threads;
  	kernelNodeParams02.sharedMemBytes = 0;
 	kernelNodeParams02.kernelParams = (void **)kernelArgs02;
 	kernelNodeParams02.extra = NULL;

	checkCudaErrors(
    hipGraphAddKernelNode(&kernelNode3, graph, &memcpyNode3,
                             1, &kernelNodeParams02));
  	nodeDependencies3.clear();
  	nodeDependencies3.push_back(kernelNode3);
	
	hipGraphNode_t kernelNode4;
	hipKernelNodeParams kernelNodeParams03 = {0};

	void* kernelArgs03[4] = {(void *)&d_input4,(void *)&d_output4, &width, &height};
	kernelNodeParams03.func = (void *)imageBlur_horizontal;
 	kernelNodeParams03.gridDim = blocks;
  	kernelNodeParams03.blockDim = threads;
  	kernelNodeParams03.sharedMemBytes = 0;
 	kernelNodeParams03.kernelParams = (void **)kernelArgs03;
 	kernelNodeParams03.extra = NULL;

	checkCudaErrors(
    hipGraphAddKernelNode(&kernelNode4, graph, &memcpyNode4,
                             1, &kernelNodeParams03));
  	nodeDependencies4.clear();
  	nodeDependencies4.push_back(kernelNode4);

	hipGraphNode_t kernelNode5;
	hipKernelNodeParams kernelNodeParams04 = {0};

	void* kernelArgs04[4] = {(void *)&d_input5,(void *)&d_output5, &width, &height};
	kernelNodeParams04.func = (void *)imageBlur_horizontal;
 	kernelNodeParams04.gridDim = blocks;
  	kernelNodeParams04.blockDim = threads;
  	kernelNodeParams04.sharedMemBytes = 0;
 	kernelNodeParams04.kernelParams = (void **)kernelArgs04;
 	kernelNodeParams04.extra = NULL;


	checkCudaErrors(
    hipGraphAddKernelNode(&kernelNode5, graph, &memcpyNode,
                             1, &kernelNodeParams04));

  	nodeDependencies5.clear();
  	nodeDependencies5.push_back(kernelNode5);
  
	hipGraphNode_t kernelNode6;
	hipKernelNodeParams kernelNodeParams05 = {0};

	void* kernelArgs05[4] = {(void *)&d_input5,(void *)&d_output5, &width, &height};
	kernelNodeParams05.func = (void *)imageBlur_horizontal;
 	kernelNodeParams05.gridDim = blocks;
  	kernelNodeParams05.blockDim = threads;
  	kernelNodeParams05.sharedMemBytes = 0;
 	kernelNodeParams05.kernelParams = (void **)kernelArgs05;
 	kernelNodeParams05.extra = NULL;

	checkCudaErrors(
    hipGraphAddKernelNode(&kernelNode6, graph, &memcpyNode6,
                             1, &kernelNodeParams05));
  	nodeDependencies6.clear();
  	nodeDependencies6.push_back(kernelNode6);

	hipGraphNode_t kernelNode7;
	hipKernelNodeParams kernelNodeParams06 = {0};

	void* kernelArgs06[4] = {(void *)&d_input7,(void *)&d_output7, &width, &height};
	kernelNodeParams06.func = (void *)imageBlur_horizontal;
 	kernelNodeParams06.gridDim = blocks;
  	kernelNodeParams06.blockDim = threads;
  	kernelNodeParams06.sharedMemBytes = 0;
 	kernelNodeParams06.kernelParams = (void **)kernelArgs06;
 	kernelNodeParams06.extra = NULL;

	checkCudaErrors(
    hipGraphAddKernelNode(&kernelNode7, graph, &memcpyNode7,
                             1, &kernelNodeParams06));
  	nodeDependencies7.clear();
  	nodeDependencies7.push_back(kernelNode7);
	
	hipGraphNode_t kernelNode8;
	hipKernelNodeParams kernelNodeParams07 = {0};

	void* kernelArgs07[4] = {(void *)&d_input8,(void *)&d_output8, &width, &height};
	kernelNodeParams07.func = (void *)imageBlur_horizontal;
 	kernelNodeParams07.gridDim = blocks;
  	kernelNodeParams07.blockDim = threads;
  	kernelNodeParams07.sharedMemBytes = 0;
 	kernelNodeParams07.kernelParams = (void **)kernelArgs07;
 	kernelNodeParams07.extra = NULL;

	checkCudaErrors(
    hipGraphAddKernelNode(&kernelNode8, graph, &memcpyNode8,
                             1, &kernelNodeParams07));
  	nodeDependencies8.clear();
  	nodeDependencies8.push_back(kernelNode8);
	

	kernelNodeParams.func = (void *)imageBlur_vertical;

	checkCudaErrors(
    hipGraphAddKernelNode(&kernelNode, graph, &memcpyNode,
                             1, &kernelNodeParams));

  	nodeDependencies.push_back(kernelNode);

	kernelNodeParams01.func = (void *)imageBlur_vertical;
	
	checkCudaErrors(
    hipGraphAddKernelNode(&kernelNode2, graph, &memcpyNode2,
                             1, &kernelNodeParams01));

  	nodeDependencies2.push_back(kernelNode2);

	
	kernelNodeParams02.func = (void *)imageBlur_vertical;
	
	checkCudaErrors(
    hipGraphAddKernelNode(&kernelNode3, graph, &memcpyNode3,
                             1, &kernelNodeParams02));

  	nodeDependencies3.push_back(kernelNode3);

	
	kernelNodeParams03.func = (void *)imageBlur_vertical;
	
	checkCudaErrors(
    hipGraphAddKernelNode(&kernelNode4, graph, &memcpyNode4,
                             1, &kernelNodeParams03));

  	nodeDependencies4.push_back(kernelNode4);

	kernelNodeParams04.func = (void *)imageBlur_vertical;

	checkCudaErrors(
    hipGraphAddKernelNode(&kernelNode5, graph, &memcpyNode5,
                             1, &kernelNodeParams04));

  	nodeDependencies.push_back(kernelNode5);

	kernelNodeParams05.func = (void *)imageBlur_vertical;
	
	checkCudaErrors(
    hipGraphAddKernelNode(&kernelNode6, graph, &memcpyNode6,
                             1, &kernelNodeParams05));

  	nodeDependencies6.push_back(kernelNode6);

	
	kernelNodeParams06.func = (void *)imageBlur_vertical;
	
	checkCudaErrors(
    hipGraphAddKernelNode(&kernelNode7, graph, &memcpyNode7,
                             1, &kernelNodeParams06));

  	nodeDependencies7.push_back(kernelNode7);

	
	kernelNodeParams07.func = (void *)imageBlur_vertical;
	
	checkCudaErrors(
    hipGraphAddKernelNode(&kernelNode8, graph, &memcpyNode8,
                             1, &kernelNodeParams07));

  	nodeDependencies8.push_back(kernelNode8);

	hipGraphNode_t empty_node;
	checkCudaErrors(
      hipGraphAddEmptyNode(&empty_node, graph, nodeDependencies.data(),
                             nodeDependencies.size()));

	hipGraphNode_t empty_node2;
	checkCudaErrors(
      hipGraphAddEmptyNode(&empty_node2, graph, nodeDependencies2.data(),
                             nodeDependencies2.size()));

	hipGraphNode_t empty_node3;
	checkCudaErrors(
      hipGraphAddEmptyNode(&empty_node3, graph, nodeDependencies3.data(),
                             nodeDependencies3.size()));

	hipGraphNode_t empty_node4;
	checkCudaErrors(
      hipGraphAddEmptyNode(&empty_node4, graph, nodeDependencies4.data(),
                             nodeDependencies4.size()));

	hipGraphNode_t empty_node5;
	checkCudaErrors(
      hipGraphAddEmptyNode(&empty_node5, graph, nodeDependencies5.data(),
                             nodeDependencies5.size()));

	hipGraphNode_t empty_node6;
	checkCudaErrors(
      hipGraphAddEmptyNode(&empty_node6, graph, nodeDependencies6.data(),
                             nodeDependencies6.size()));

	hipGraphNode_t empty_node7;
	checkCudaErrors(
      hipGraphAddEmptyNode(&empty_node7, graph, nodeDependencies7.data(),
                             nodeDependencies7.size()));

	hipGraphNode_t empty_node8;
	checkCudaErrors(
      hipGraphAddEmptyNode(&empty_node8, graph, nodeDependencies8.data(),
                             nodeDependencies8.size()));

	void* kernelArgs2[4] = {(void *)&d_input, (void *)&gradient_h_output, &width, &height};
	kernelNodeParams.func = (void *)gradient_horizontal;
 	kernelNodeParams.kernelParams = (void **)kernelArgs2;
	
	checkCudaErrors(
    hipGraphAddKernelNode(&kernelNode, graph, &empty_node,
                             1, &kernelNodeParams));

  	nodeDependencies.clear();
  	nodeDependencies.push_back(kernelNode);

	void* kernelArgs21[4] = {(void *)&d_input2, (void *)&gradient_h_output2, &width, &height};
	kernelNodeParams01.func = (void *)gradient_horizontal;
 	kernelNodeParams01.kernelParams = (void **)kernelArgs21;
	
	checkCudaErrors(
    hipGraphAddKernelNode(&kernelNode2, graph, &empty_node2,
                             1, &kernelNodeParams01));

  	nodeDependencies2.clear();
  	nodeDependencies2.push_back(kernelNode2);


	void* kernelArgs22[4] = {(void *)&d_input3, (void *)&gradient_h_output3, &width, &height};
	kernelNodeParams02.func = (void *)gradient_horizontal;
 	kernelNodeParams02.kernelParams = (void **)kernelArgs22;
	
	checkCudaErrors(
    hipGraphAddKernelNode(&kernelNode3, graph, &empty_node3,
                             1, &kernelNodeParams02));

  	nodeDependencies3.clear();
  	nodeDependencies3.push_back(kernelNode3);

	
	void* kernelArgs23[4] = {(void *)&d_input4, (void *)&gradient_h_output4, &width, &height};
	kernelNodeParams03.func = (void *)gradient_horizontal;
 	kernelNodeParams03.kernelParams = (void **)kernelArgs23;
	
	checkCudaErrors(
    hipGraphAddKernelNode(&kernelNode4, graph, &empty_node4,
                             1, &kernelNodeParams03));

  	nodeDependencies4.clear();
  	nodeDependencies4.push_back(kernelNode4);
	
	void* kernelArgs24[4] = {(void *)&d_input5, (void *)&gradient_h_output5, &width, &height};
	kernelNodeParams04.func = (void *)gradient_horizontal;
 	kernelNodeParams04.kernelParams = (void **)kernelArgs24;
	
	checkCudaErrors(
    hipGraphAddKernelNode(&kernelNode5, graph, &empty_node5,
                             1, &kernelNodeParams04));

  	nodeDependencies5.clear();
  	nodeDependencies5.push_back(kernelNode5);

	void* kernelArgs25[4] = {(void *)&d_input5, (void *)&gradient_h_output5, &width, &height};
	kernelNodeParams05.func = (void *)gradient_horizontal;
 	kernelNodeParams05.kernelParams = (void **)kernelArgs25;
	
	checkCudaErrors(
    hipGraphAddKernelNode(&kernelNode6, graph, &empty_node6,
                             1, &kernelNodeParams05));

  	nodeDependencies6.clear();
  	nodeDependencies6.push_back(kernelNode6);


	void* kernelArgs26[4] = {(void *)&d_input7, (void *)&gradient_h_output7, &width, &height};
	kernelNodeParams06.func = (void *)gradient_horizontal;
 	kernelNodeParams06.kernelParams = (void **)kernelArgs26;
	
	checkCudaErrors(
    hipGraphAddKernelNode(&kernelNode7, graph, &empty_node7,
                             1, &kernelNodeParams06));

  	nodeDependencies7.clear();
  	nodeDependencies7.push_back(kernelNode7);

	
	void* kernelArgs27[4] = {(void *)&d_input8, (void *)&gradient_h_output8, &width, &height};
	kernelNodeParams07.func = (void *)gradient_horizontal;
 	kernelNodeParams07.kernelParams = (void **)kernelArgs27;
	
	checkCudaErrors(
    hipGraphAddKernelNode(&kernelNode8, graph, &empty_node8,
                             1, &kernelNodeParams07));

  	nodeDependencies4.clear();
  	nodeDependencies4.push_back(kernelNode8);
	
	
	void* kernelArgs3[4] = {(void *)&d_input,(void *)& gradient_v_output, &width, &height};
	kernelNodeParams.func = (void *)gradient_vertical;
 	kernelNodeParams.kernelParams = (void **)kernelArgs3;

	checkCudaErrors(
    hipGraphAddKernelNode(&kernelNode, graph, &empty_node,
                             1, &kernelNodeParams));

  	nodeDependencies.push_back(kernelNode);

	void* kernelArgs31[4] = {(void *)&d_input2,(void *)& gradient_v_output2, &width, &height};
	kernelNodeParams01.func = (void *)gradient_vertical;
 	kernelNodeParams01.kernelParams = (void **)kernelArgs31;

	checkCudaErrors(
    hipGraphAddKernelNode(&kernelNode2, graph, &empty_node2,
                             1, &kernelNodeParams01));

  	nodeDependencies2.push_back(kernelNode2);

	
	void* kernelArgs32[4] = {(void *)&d_input3,(void *)& gradient_v_output3, &width, &height};
	kernelNodeParams02.func = (void *)gradient_vertical;
 	kernelNodeParams02.kernelParams = (void **)kernelArgs32;

	checkCudaErrors(
    hipGraphAddKernelNode(&kernelNode3, graph, &empty_node3,
                             1, &kernelNodeParams02));

  	nodeDependencies3.push_back(kernelNode3);


	void* kernelArgs33[4] = {(void *)&d_input4,(void *)& gradient_v_output4, &width, &height};
	kernelNodeParams03.func = (void *)gradient_vertical;
 	kernelNodeParams03.kernelParams = (void **)kernelArgs33;

	checkCudaErrors(
    hipGraphAddKernelNode(&kernelNode4, graph, &empty_node4,
                             1, &kernelNodeParams03));

  	nodeDependencies4.push_back(kernelNode4);

	void* kernelArgs34[4] = {(void *)&d_input5,(void *)& gradient_v_output5, &width, &height};
	kernelNodeParams04.func = (void *)gradient_vertical;
 	kernelNodeParams04.kernelParams = (void **)kernelArgs04;

	checkCudaErrors(
    hipGraphAddKernelNode(&kernelNode5, graph, &empty_node5,
                             1, &kernelNodeParams04));

  	nodeDependencies.push_back(kernelNode5);

	void* kernelArgs35[4] = {(void *)&d_input6,(void *)& gradient_v_output6, &width, &height};
	kernelNodeParams05.func = (void *)gradient_vertical;
 	kernelNodeParams05.kernelParams = (void **)kernelArgs35;

	checkCudaErrors(
    hipGraphAddKernelNode(&kernelNode6, graph, &empty_node6,
                             1, &kernelNodeParams05));

  	nodeDependencies6.push_back(kernelNode6);

	
	void* kernelArgs36[4] = {(void *)&d_input7,(void *)& gradient_v_output7, &width, &height};
	kernelNodeParams06.func = (void *)gradient_vertical;
 	kernelNodeParams06.kernelParams = (void **)kernelArgs36;

	checkCudaErrors(
    hipGraphAddKernelNode(&kernelNode7, graph, &empty_node7,
                             1, &kernelNodeParams06));

  	nodeDependencies7.push_back(kernelNode7);


	void* kernelArgs37[4] = {(void *)&d_input8,(void *)& gradient_v_output8, &width, &height};
	kernelNodeParams07.func = (void *)gradient_vertical;
 	kernelNodeParams07.kernelParams = (void **)kernelArgs37;

	checkCudaErrors(
    hipGraphAddKernelNode(&kernelNode8, graph, &empty_node8,
                             1, &kernelNodeParams07));

  	nodeDependencies8.push_back(kernelNode8);


	void* kernelArgs4[6] = {(void *)&d_input, (void *)&d_output, (void *)&gradient_h_output, (void *)&gradient_v_output, &width, &height};
	kernelNodeParams.func = (void *)sobelFilter;
 	kernelNodeParams.kernelParams = (void **)kernelArgs4;

	checkCudaErrors(
    hipGraphAddKernelNode(&kernelNode, graph, nodeDependencies.data(),
                             nodeDependencies.size(), &kernelNodeParams));

  	nodeDependencies.clear();
  	nodeDependencies.push_back(kernelNode);

	void* kernelArgs41[6] = {(void *)&d_input2, (void *)&d_output2, (void *)&gradient_h_output2, (void *)&gradient_v_output2, &width, &height};
	kernelNodeParams01.func = (void *)sobelFilter;
 	kernelNodeParams01.kernelParams = (void **)kernelArgs41;

	checkCudaErrors(
    hipGraphAddKernelNode(&kernelNode2, graph, nodeDependencies2.data(),
                             nodeDependencies2.size(), &kernelNodeParams01));

  	nodeDependencies2.clear();
  	nodeDependencies2.push_back(kernelNode2);

	void* kernelArgs42[6] = {(void *)&d_input3, (void *)&d_output3, (void *)&gradient_h_output3, (void *)&gradient_v_output3, &width, &height};
	kernelNodeParams02.func = (void *)sobelFilter;
 	kernelNodeParams02.kernelParams = (void **)kernelArgs42;

	checkCudaErrors(
    hipGraphAddKernelNode(&kernelNode3, graph, nodeDependencies3.data(),
                             nodeDependencies3.size(), &kernelNodeParams02));

  	nodeDependencies3.clear();
  	nodeDependencies3.push_back(kernelNode3);


	void* kernelArgs43[6] = {(void *)&d_input4, (void *)&d_output4, (void *)&gradient_h_output4, (void *)&gradient_v_output4, &width, &height};
	kernelNodeParams03.func = (void *)sobelFilter;
 	kernelNodeParams03.kernelParams = (void **)kernelArgs43;

	checkCudaErrors(
    hipGraphAddKernelNode(&kernelNode4, graph, nodeDependencies4.data(),
                             nodeDependencies4.size(), &kernelNodeParams03));

  	nodeDependencies4.clear();
  	nodeDependencies4.push_back(kernelNode4);
	
	void* kernelArgs44[6] = {(void *)&d_input5, (void *)&d_output5, (void *)&gradient_h_output5, (void *)&gradient_v_output5, &width, &height};
	kernelNodeParams04.func = (void *)sobelFilter;
 	kernelNodeParams04.kernelParams = (void **)kernelArgs44;

	checkCudaErrors(
    hipGraphAddKernelNode(&kernelNode5, graph, nodeDependencies5.data(),
                             nodeDependencies5.size(), &kernelNodeParams04));

  	nodeDependencies5.clear();
  	nodeDependencies5.push_back(kernelNode5);

	void* kernelArgs45[6] = {(void *)&d_input6, (void *)&d_output6, (void *)&gradient_h_output6, (void *)&gradient_v_output6, &width, &height};
	kernelNodeParams05.func = (void *)sobelFilter;
 	kernelNodeParams05.kernelParams = (void **)kernelArgs45;

	checkCudaErrors(
    hipGraphAddKernelNode(&kernelNode6, graph, nodeDependencies6.data(),
                             nodeDependencies6.size(), &kernelNodeParams05));

  	nodeDependencies6.clear();
  	nodeDependencies6.push_back(kernelNode6);

	void* kernelArgs46[6] = {(void *)&d_input7, (void *)&d_output7, (void *)&gradient_h_output7, (void *)&gradient_v_output7, &width, &height};
	kernelNodeParams06.func = (void *)sobelFilter;
 	kernelNodeParams06.kernelParams = (void **)kernelArgs46;

	checkCudaErrors(
    hipGraphAddKernelNode(&kernelNode7, graph, nodeDependencies7.data(),
                             nodeDependencies7.size(), &kernelNodeParams06));

  	nodeDependencies7.clear();
  	nodeDependencies7.push_back(kernelNode7);


	void* kernelArgs47[6] = {(void *)&d_input8, (void *)&d_output8, (void *)&gradient_h_output8, (void *)&gradient_v_output8, &width, &height};
	kernelNodeParams07.func = (void *)sobelFilter;
 	kernelNodeParams07.kernelParams = (void **)kernelArgs47;

	checkCudaErrors(
    hipGraphAddKernelNode(&kernelNode8, graph, nodeDependencies8.data(),
                             nodeDependencies8.size(), &kernelNodeParams07));

  	nodeDependencies8.clear();
  	nodeDependencies8.push_back(kernelNode8);

	memcpyParams.srcPtr = make_hipPitchedPtr(d_output, memSize, 1, 1);
	memcpyParams.dstPtr = make_hipPitchedPtr(final, memSize, 1, 1);
	memcpyParams.extent = make_hipExtent(memSize, 1, 1);
	memcpyParams.kind = hipMemcpyDeviceToHost;

	checkCudaErrors(
      hipGraphAddMemcpyNode(&memcpyNode, graph, &kernelNode,
                             1, &memcpyParams));


	memcpyParams01.srcPtr = make_hipPitchedPtr(d_output2, memSize, 1, 1);
	memcpyParams01.dstPtr = make_hipPitchedPtr(final2, memSize, 1, 1);
	memcpyParams01.extent = make_hipExtent(memSize, 1, 1);
	memcpyParams01.kind = hipMemcpyDeviceToHost;

	checkCudaErrors(
      hipGraphAddMemcpyNode(&memcpyNode2, graph, &kernelNode2,
                             1, &memcpyParams01));
	
	memcpyParams02.srcPtr = make_hipPitchedPtr(d_output3, memSize, 1, 1);
	memcpyParams02.dstPtr = make_hipPitchedPtr(final3, memSize, 1, 1);
	memcpyParams02.extent = make_hipExtent(memSize, 1, 1);
	memcpyParams02.kind = hipMemcpyDeviceToHost;

	checkCudaErrors(
      hipGraphAddMemcpyNode(&memcpyNode3, graph, &kernelNode3,
                             1, &memcpyParams02));

	memcpyParams03.srcPtr = make_hipPitchedPtr(d_output4, memSize, 1, 1);
	memcpyParams03.dstPtr = make_hipPitchedPtr(final4, memSize, 1, 1);
	memcpyParams03.extent = make_hipExtent(memSize, 1, 1);
	memcpyParams03.kind = hipMemcpyDeviceToHost;

	checkCudaErrors(
      hipGraphAddMemcpyNode(&memcpyNode4, graph, &kernelNode4,
                             1, &memcpyParams03));

	memcpyParams04.srcPtr = make_hipPitchedPtr(d_output5, memSize, 1, 1);
	memcpyParams04.dstPtr = make_hipPitchedPtr(final5, memSize, 1, 1);
	memcpyParams04.extent = make_hipExtent(memSize, 1, 1);
	memcpyParams04.kind = hipMemcpyDeviceToHost;

	checkCudaErrors(
      hipGraphAddMemcpyNode(&memcpyNode, graph, &kernelNode,
                             1, &memcpyParams));


	memcpyParams05.srcPtr = make_hipPitchedPtr(d_output6, memSize, 1, 1);
	memcpyParams05.dstPtr = make_hipPitchedPtr(final6, memSize, 1, 1);
	memcpyParams05.extent = make_hipExtent(memSize, 1, 1);
	memcpyParams05.kind = hipMemcpyDeviceToHost;

	checkCudaErrors(
      hipGraphAddMemcpyNode(&memcpyNode6, graph, &kernelNode6,
                             1, &memcpyParams05));
	
	memcpyParams06.srcPtr = make_hipPitchedPtr(d_output7, memSize, 1, 1);
	memcpyParams06.dstPtr = make_hipPitchedPtr(final7, memSize, 1, 1);
	memcpyParams06.extent = make_hipExtent(memSize, 1, 1);
	memcpyParams06.kind = hipMemcpyDeviceToHost;

	checkCudaErrors(
      hipGraphAddMemcpyNode(&memcpyNode7, graph, &kernelNode7,
                             1, &memcpyParams06));

	memcpyParams07.srcPtr = make_hipPitchedPtr(d_output8, memSize, 1, 1);
	memcpyParams07.dstPtr = make_hipPitchedPtr(final8, memSize, 1, 1);
	memcpyParams07.extent = make_hipExtent(memSize, 1, 1);
	memcpyParams07.kind = hipMemcpyDeviceToHost;

	checkCudaErrors(
      hipGraphAddMemcpyNode(&memcpyNode8, graph, &kernelNode8,
                             1, &memcpyParams07));


	checkCudaErrors(hipGraphDebugDotPrint(graph, "mainGraph.dot", 0));


	hipGraphExec_t graphExec;
  	checkCudaErrors(hipGraphInstantiate(&graphExec, graph, NULL, NULL, 0));
	
	warm_up_gpu << <blocks, threads >> > ();

	hipEventCreate(&start_sobel);
  	hipEventCreate(&stop_sobel);

    hipEventRecord(start_sobel, 0);
	
	checkCudaErrors(hipGraphLaunch(graphExec, 0));
	checkCudaErrors(hipStreamSynchronize(0));

	hipEventRecord(stop_sobel, 0);
  	hipEventSynchronize(stop_sobel);
  	hipEventElapsedTime(&sobel, start_sobel, stop_sobel);

	printf("Device Time:  %f s \n", sobel/1000);

	hipError_t err = hipGetLastError();
	if (hipSuccess != err)
	{
		fprintf(stderr, "Cuda error: %s: %s.\n", "Main Loop", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	checkCudaErrors(hipGraphExecDestroy(graphExec));
  	checkCudaErrors(hipGraphDestroy(graph));
	hipFree(d_input);
	hipFree(d_output);
	hipFree(gradient_h_output);
	hipFree(gradient_v_output);

	hipFree(d_input2);
	hipFree(d_output2);
	hipFree(gradient_h_output2);
	hipFree(gradient_v_output2);

	hipFree(d_input3);
	hipFree(d_output3);
	hipFree(gradient_h_output3);
	hipFree(gradient_v_output3);

	hipFree(d_input4);
	hipFree(d_output4);
	hipFree(gradient_h_output4);
	hipFree(gradient_v_output4);

	hipFree(d_input5);
	hipFree(d_output5);
	hipFree(gradient_h_output5);
	hipFree(gradient_v_output5);

	hipFree(d_input6);
	hipFree(d_output6);
	hipFree(gradient_h_output6);
	hipFree(gradient_v_output6);

	hipFree(d_input7);
	hipFree(d_output7);
	hipFree(gradient_h_output7);
	hipFree(gradient_v_output7);

	hipFree(d_input8);
	hipFree(d_output8);
	hipFree(gradient_h_output8);
	hipFree(gradient_v_output8);

   
  hipEventRecord(stop_total, 0);
  hipEventSynchronize(stop_total);
  hipEventElapsedTime(&total, start_total, stop_total);

  printf("Total Time:  %f s \n", total/1000);
  
	// write image
	pgmwrite("../images/image-output_g_apollonian_gasket.ascii.pgm", (void *)final,width, height);
	pgmwrite("../images/image-output_2g_apollonian_gasket.ascii.pgm", (void *)final2,width, height);
	pgmwrite("../images/image-output_3g_apollonian_gasket.ascii.pgm", (void *)final3,width, height);
	pgmwrite("../images/image-output_4g_apollonian_gasket.ascii.pgm", (void *)final4,width, height);
	pgmwrite("../images/image-output_5g_apollonian_gasket.ascii.pgm", (void *)final5,width, height);
	pgmwrite("../images/image-output_6g_apollonian_gasket.ascii.pgm", (void *)final6,width, height);
	pgmwrite("../images/image-output_7g_apollonian_gasket.ascii.pgm", (void *)final7,width, height);
	pgmwrite("../images/image-output_8g_apollonian_gasket.ascii.pgm", (void *)final8,width, height);
	hipDeviceReset();
	
	return 0;


}

