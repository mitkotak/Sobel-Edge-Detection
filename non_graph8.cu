#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#ifdef __INTELLISENSE__
void __syncthreads();
#endif

#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>
#include <pgmio.h>
#include <vector>

// image dimensions WIDTH & HEIGHT
#define WIDTH 600
#define HEIGHT 600

// Block width WIDTH & HEIGHT
#define BLOCK_W 10
#define BLOCK_H 10

// buffer to read image into
// float image[HEIGHT][WIDTH];

// buffer for resulting image
// float final[HEIGHT][WIDTH];


#define MAXLINE 128

float total, sobel;
hipEvent_t start_total, stop_total;
hipEvent_t start_sobel, stop_sobel;

__global__ void warm_up_gpu(){
  unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
  float ia, ib;
  ia = ib = 0.0f;
  ib += ia + tid; 
}

__global__ void imageBlur_horizontal(float *input, float *output, int width, int height) {

	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;

	int numcols = WIDTH;

	float blur;

	if (row <= height && col <= width && row > 0 && col > 0)
	{
		// weights
		int	x3, x4, x5;

		// blur
		// 0.2 0.2 0.2

		x3 = input[row * numcols + (col - 1)];			// left
		x4 = input[row * numcols + col];				// center
		x5 = input[row * numcols + (col + 1)];			// right

		blur =  (x3 * 0.2) + (x4 * 0.2) + (x5 * 0.2);

		output[row * numcols + col] = blur;
	}
}

__global__ void imageBlur_vertical(float *input, float *output, int width, int height) {
	
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;

	int numcols = WIDTH;

	float blur;

	if (row <= height && col <= width && row > 0 && col > 0)
	{
		// weights
		int		x1,x7;

		// blur
		// 0.0 0.2 0.0
		// 0.2 0.2 0.2
		// 0.0 0.2 0.0

		x1 = input[(row + 1) * numcols + col];			// up
		x7 = input[(row + -1) * numcols + col];			// down

		blur = (x1 * 0.2) + (x7 * 0.2);

		output[row * numcols + col] = blur;
	}
}


__global__ void gradient_horizontal(float *input, float *output, int width, int height) {

	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;

	int numcols = WIDTH;


	// horizontal
	// -1  0  1
	// -2  0  2
	// -1  0  1

	int x0, x2, 
		x3, x5, 
		x6, x8;

	if (row <= height && col <= width && row > 0 && col > 0)
	{
	
	x0 = input[(row - 1) * numcols + (col - 1)];	// leftup
	x2 = input[(row - 1) * numcols + (col + 1)];	// rightup
	x3 = input[row * numcols + (col - 1)];			// left
	x5 = input[row * numcols + (col + 1)];			// right
	x6 = input[(row + 1) * numcols + (col - 1)];	// leftdown
	x8 = input[(row + 1) * numcols + (col + 1)];	// rightdown


	output[row * numcols + col] = (x0 * -1) + (x2 * 1) + (x3 * -2) + (x5 * 2) + (x6 * -1) + (x8 * 1);

	}

	return;
}


__global__ void gradient_vertical(float *input, float *output, int width, int height) {

	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;

	int numcols = WIDTH;

	// vertical 
	// -1 -2 -1
	//  0  0  0
	//  1  2  1

	int x0, x1, 
		x3, x6, 
		x7, x8;

	if (row <= height && col <= width && row > 0 && col > 0)
	{
		x0 = input[(row - 1) * numcols + (col - 1)];	// leftup
		x1 = input[(row + 1) * numcols + col];			// up
		x3 = input[row * numcols + (col - 1)];			// left
		x6 = input[(row + 1) * numcols + (col - 1)];	// leftdown
		x7 = input[(row + -1) * numcols + col];			// down
		x8 = input[(row + 1) * numcols + (col + 1)];	// rightdown


		output[row * numcols + col] = (x0 * -1) + (x1 * -2) + (x3 * -1) + (x6 * 1) + (x7 * 2) + (x8 * 1);

	}
	
}

__global__ void sobelFilter(float *input, float *output, float *gradient_h_output, float *gradient_v_output, int width, int height) {
	
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;

	int numcols = WIDTH;

	float gradient,gradient_h,gradient_v;	
	float thresh = 30;	

	if (row <= height && col <= width && row > 0 && col > 0)
	{	

		gradient_h = gradient_h_output[row * numcols + col];
		gradient_v = gradient_v_output[row * numcols + col];
		gradient = sqrt((gradient_h * gradient_h) + (gradient_v * gradient_v));

		if (gradient >= thresh)
		{
			gradient = 255;
		}
		else {
			gradient = 0;
		}
		output[row * numcols + col] = gradient;
	}
}

	
	// pgmread("image100000x100000.pgm", (void *)image, WIDTH, HEIGHT);
	// pgmread("../images/image20000x20000.pgm", (void *)image, WIDTH, HEIGHT);
	// pgmread("../images/image16384x16384.pgm", (void *)image, WIDTH, HEIGHT);
	// pgmread("image10000x10000.pgm", (void *)image, WIDTH, HEIGHT);
	// pgmread("../images/image4096x4096.pgm", (void *)image, WIDTH, HEIGHT);
	// pgmread("../images/image2048x2048.pgm", (void *)image, WIDTH, HEIGHT);
	// pgmread("../images/rabbit2000x3000.pgm", (void *)image, WIDTH, HEIGHT);
	// pgmwrite("../images/image1024x1024.pgm", (void *)image, WIDTH, HEIGHT);
	// pgmread("../images/image512x512.pgm", (void *)image, WIDTH, HEIGHT);
	// pgmread("pgmimg.pgm", (void *)image, WIDTH, HEIGHT);

	// pgmwrite("image-outputl100000x100000.pgm", (void *)final, WIDTH, HEIGHT);
	// pgmwrite("image-output_ng_20000x20000.pgm", (void *)final, WIDTH, HEIGHT);
	// pgmwrite("../images/image-output_ng_16384x16384.pgm", (void *)final, WIDTH, HEIGHT);
	// pgmwrite("image-outputl10000x10000.pgm", (void *)final, WIDTH, HEIGHT);
	// pgmwrite("../images/image-output_g_4096x4096.pgm", (void *)final, WIDTH, HEIGHT);
	// pgmwrite("../images/image-output_ng_2048x2048.pgm", (void *)final, WIDTH, HEIGHT);
	// pgmwrite("../images/image-output_ng_2000x3000.pgm", (void *)final, WIDTH, HEIGHT);
	// pgmwrite("../images/image-output_ng_1024x1024.pgm", (void *)final, WIDTH, HEIGHT);
	// pgmwrite("../images/image-output_ng_512x512.pgm", (void *)final, WIDTH, HEIGHT);
	// // pgmwrite("pgmimg-output.pgm", (void *)final, WIDTH, HEIGHT);


int main(int argc, char *argv[])
{ int width = 600, height=600;
  float *image = NULL, *final = NULL;
  float *image2 = NULL, *final2 = NULL;
  float *image3 = NULL, *final3 = NULL;
  float *image4 = NULL, *final4 = NULL;
  float *image5 = NULL, *final5 = NULL;
  float *image6 = NULL, *final6 = NULL;
  float *image7 = NULL, *final7 = NULL;
  float *image8 = NULL, *final8 = NULL;

  size_t memSize = width * height * sizeof(float);
  checkCudaErrors((hipHostMalloc(&image, memSize)));
  checkCudaErrors((hipHostMalloc(&final, memSize)));

 checkCudaErrors((hipHostMalloc(&image2, memSize)));
  checkCudaErrors((hipHostMalloc(&final2, memSize)));

checkCudaErrors((hipHostMalloc(&image3, memSize)));
  checkCudaErrors((hipHostMalloc(&final3, memSize)));

 checkCudaErrors((hipHostMalloc(&image4, memSize)));
  checkCudaErrors((hipHostMalloc(&final4, memSize)));

  checkCudaErrors((hipHostMalloc(&image5, memSize)));
  checkCudaErrors((hipHostMalloc(&final5, memSize)));

 checkCudaErrors((hipHostMalloc(&image6, memSize)));
  checkCudaErrors((hipHostMalloc(&final6, memSize)));

checkCudaErrors((hipHostMalloc(&image7, memSize)));
  checkCudaErrors((hipHostMalloc(&final7, memSize)));

 checkCudaErrors((hipHostMalloc(&image8, memSize)));
  checkCudaErrors((hipHostMalloc(&final8, memSize)));

  // read image 
  pgmread("../images/test_images/apollonian_gasket.ascii.pgm", (void *)image, width, height);
  pgmread("../images/test_images/apollonian_gasket.ascii.pgm", (void *)image2, width, height);
  pgmread("../images/test_images/apollonian_gasket.ascii.pgm", (void *)image3, width, height);
  pgmread("../images/test_images/apollonian_gasket.ascii.pgm", (void *)image4, width, height);
   pgmread("../images/test_images/apollonian_gasket.ascii.pgm", (void *)image5, width, height);
  pgmread("../images/test_images/apollonian_gasket.ascii.pgm", (void *)image6, width, height);
  pgmread("../images/test_images/apollonian_gasket.ascii.pgm", (void *)image7, width, height);
  pgmread("../images/test_images/apollonian_gasket.ascii.pgm", (void *)image8, width, height);
  
  hipEventCreate(&start_total);
  hipEventCreate(&start_total);
  hipEventCreate(&start_total);
  hipEventCreate(&start_total);
  hipEventCreate(&stop_total);
  hipEventRecord(start_total, 0);

	int x, y;
	float *d_input, *d_output, *gradient_h_output, *gradient_v_output;
	float *d_input2, *d_output2, *gradient_h_output2, *gradient_v_output2;
	float *d_input3, *d_output3, *gradient_h_output3, *gradient_v_output3;
	float *d_input4, *d_output4, *gradient_h_output4, *gradient_v_output4;
	float *d_input5, *d_output5, *gradient_h_output5, *gradient_v_output5;
	float *d_input6, *d_output6, *gradient_h_output6, *gradient_v_output6;
	float *d_input7, *d_output7, *gradient_h_output7, *gradient_v_output7;
	float *d_input8, *d_output8, *gradient_h_output8, *gradient_v_output8;


	printf("Block size: %dx%d\n", BLOCK_W, BLOCK_H);

	printf("Blocks per grid (width): %d |", (width / BLOCK_W));
	printf("Blocks per grid (height): %d \n", (height / BLOCK_H));
	
	dim3 threads(BLOCK_W, BLOCK_H); // threads per block
	dim3 blocks(width / BLOCK_W, height / BLOCK_H); // blocks per grid 

	warm_up_gpu << <blocks, threads >> > ();

	hipMalloc(&d_input, memSize);
	hipMalloc(&d_output, memSize);
	hipMalloc(&gradient_h_output, memSize);
	hipMalloc(&gradient_v_output, memSize);

	hipMalloc(&d_input2, memSize);
	hipMalloc(&d_output2, memSize);
	hipMalloc(&gradient_h_output2, memSize);
	hipMalloc(&gradient_v_output2, memSize);

	hipMalloc(&d_input3, memSize);
	hipMalloc(&d_output3, memSize);
	hipMalloc(&gradient_h_output3, memSize);
	hipMalloc(&gradient_v_output3, memSize);

	hipMalloc(&d_input4, memSize);
	hipMalloc(&d_output4, memSize);
	hipMalloc(&gradient_h_output4, memSize);
	hipMalloc(&gradient_v_output4, memSize);

		hipMalloc(&d_input5, memSize);
	hipMalloc(&d_output5, memSize);
	hipMalloc(&gradient_h_output5, memSize);
	hipMalloc(&gradient_v_output5, memSize);

	hipMalloc(&d_input6, memSize);
	hipMalloc(&d_output6, memSize);
	hipMalloc(&gradient_h_output6, memSize);
	hipMalloc(&gradient_v_output6, memSize);

	hipMalloc(&d_input7, memSize);
	hipMalloc(&d_output7, memSize);
	hipMalloc(&gradient_h_output7, memSize);
	hipMalloc(&gradient_v_output7, memSize);

	hipMalloc(&d_input8, memSize);
	hipMalloc(&d_output8, memSize);
	hipMalloc(&gradient_h_output8, memSize);
	hipMalloc(&gradient_v_output8, memSize);

	hipEventCreate(&start_sobel);
  	hipEventCreate(&stop_sobel);

	hipEventRecord(start_sobel, 0);

	hipMemcpy(d_input, image, memSize, hipMemcpyHostToDevice);

	hipMemcpy(d_input2, image2, memSize, hipMemcpyHostToDevice);

	hipMemcpy(d_input3, image3, memSize, hipMemcpyHostToDevice);

	hipMemcpy(d_input4, image4, memSize, hipMemcpyHostToDevice);

	hipMemcpy(d_input5, image5, memSize, hipMemcpyHostToDevice);

	hipMemcpy(d_input6, image6, memSize, hipMemcpyHostToDevice);

	hipMemcpy(d_input7, image7, memSize, hipMemcpyHostToDevice);

	hipMemcpy(d_input8, image8, memSize, hipMemcpyHostToDevice);
	
	// printf("Launching imageBlur_horizontal \n");
  	imageBlur_horizontal << <blocks, threads >> > (d_input, d_output, WIDTH, HEIGHT);
	  
	imageBlur_vertical << <blocks, threads >> > (d_input, d_output, WIDTH, HEIGHT);

	hipDeviceSynchronize();
	
	imageBlur_horizontal << <blocks, threads >> > (d_input2, d_output2, WIDTH, HEIGHT);

	imageBlur_vertical << <blocks, threads >> > (d_input2, d_output2, WIDTH, HEIGHT);

	hipDeviceSynchronize();

	imageBlur_horizontal << <blocks, threads >> > (d_input3, d_output3, WIDTH, HEIGHT);
	
	imageBlur_vertical << <blocks, threads >> > (d_input3, d_output3, WIDTH, HEIGHT);
	
	hipDeviceSynchronize();

	imageBlur_horizontal << <blocks, threads >> > (d_input4, d_output4, WIDTH, HEIGHT);
	
	imageBlur_vertical << <blocks, threads >> > (d_input4, d_output4, WIDTH, HEIGHT);
	
  	hipDeviceSynchronize();

	  	imageBlur_horizontal << <blocks, threads >> > (d_input5, d_output5, WIDTH, HEIGHT);
	  
	imageBlur_vertical << <blocks, threads >> > (d_input5, d_output5, WIDTH, HEIGHT);

	hipDeviceSynchronize();
	
	imageBlur_horizontal << <blocks, threads >> > (d_input6, d_output6, WIDTH, HEIGHT);

	imageBlur_vertical << <blocks, threads >> > (d_input6, d_output6, WIDTH, HEIGHT);

	hipDeviceSynchronize();

	imageBlur_horizontal << <blocks, threads >> > (d_input7, d_output7, WIDTH, HEIGHT);
	
	imageBlur_vertical << <blocks, threads >> > (d_input7, d_output7, WIDTH, HEIGHT);
	
	hipDeviceSynchronize();

	imageBlur_horizontal << <blocks, threads >> > (d_input8, d_output8, WIDTH, HEIGHT);
	
	imageBlur_vertical << <blocks, threads >> > (d_input8, d_output8, WIDTH, HEIGHT);
	
  	hipDeviceSynchronize();
	
	// printf("Launching gradient_horizontal \n");
	gradient_horizontal<< <blocks, threads>> >(d_input, gradient_h_output, WIDTH, HEIGHT);
	// printf("Launching gradient_vertical \n");
	gradient_vertical<< <blocks, threads>> >(d_input, gradient_v_output, WIDTH, HEIGHT);
	// printf("Launching sobelFilter \n");	
	sobelFilter << <blocks, threads >> > (d_input, d_output, gradient_h_output, gradient_v_output, WIDTH, HEIGHT);

	hipDeviceSynchronize();

	// printf("Launching gradient_horizontal \n");
	gradient_horizontal<< <blocks, threads>> >(d_input2, gradient_h_output2, WIDTH, HEIGHT);
	// printf("Launching gradient_vertical \n");
	gradient_vertical<< <blocks, threads>> >(d_input2, gradient_v_output2, WIDTH, HEIGHT);
	// printf("Launching sobelFilter \n");	
	sobelFilter << <blocks, threads >> > (d_input2, d_output2, gradient_h_output2, gradient_v_output2, WIDTH, HEIGHT);

	hipDeviceSynchronize();

	// printf("Launching gradient_horizontal \n");
	gradient_horizontal<< <blocks, threads>> >(d_input3, gradient_h_output3, WIDTH, HEIGHT);
	// printf("Launching gradient_vertical \n");
	gradient_vertical<< <blocks, threads>> >(d_input3, gradient_v_output3, WIDTH, HEIGHT);
	// printf("Launching sobelFilter \n");	
	sobelFilter << <blocks, threads >> > (d_input3, d_output3, gradient_h_output3, gradient_v_output3, WIDTH, HEIGHT);

	hipDeviceSynchronize();

		// printf("Launching gradient_horizontal \n");
	gradient_horizontal<< <blocks, threads>> >(d_input4, gradient_h_output4, WIDTH, HEIGHT);
	// printf("Launching gradient_vertical \n");
	gradient_vertical<< <blocks, threads>> >(d_input4, gradient_v_output4, WIDTH, HEIGHT);
	// printf("Launching sobelFilter \n");	
	sobelFilter << <blocks, threads >> > (d_input4, d_output4, gradient_h_output4, gradient_v_output4, WIDTH, HEIGHT);

	hipDeviceSynchronize();
	// printf("Launching gradient_horizontal \n");
	gradient_horizontal<< <blocks, threads>> >(d_input, gradient_h_output, WIDTH, HEIGHT);
	// printf("Launching gradient_vertical \n");
	gradient_vertical<< <blocks, threads>> >(d_input, gradient_v_output, WIDTH, HEIGHT);
	// printf("Launching sobelFilter \n");	
	sobelFilter << <blocks, threads >> > (d_input, d_output, gradient_h_output, gradient_v_output, WIDTH, HEIGHT);

	hipDeviceSynchronize();

	// printf("Launching gradient_horizontal \n");
	gradient_horizontal<< <blocks, threads>> >(d_input2, gradient_h_output2, WIDTH, HEIGHT);
	// printf("Launching gradient_vertical \n");
	gradient_vertical<< <blocks, threads>> >(d_input2, gradient_v_output2, WIDTH, HEIGHT);
	// printf("Launching sobelFilter \n");	
	sobelFilter << <blocks, threads >> > (d_input2, d_output2, gradient_h_output2, gradient_v_output2, WIDTH, HEIGHT);

	hipDeviceSynchronize();

	// printf("Launching gradient_horizontal \n");
	gradient_horizontal<< <blocks, threads>> >(d_input3, gradient_h_output3, WIDTH, HEIGHT);
	// printf("Launching gradient_vertical \n");
	gradient_vertical<< <blocks, threads>> >(d_input3, gradient_v_output3, WIDTH, HEIGHT);
	// printf("Launching sobelFilter \n");	
	sobelFilter << <blocks, threads >> > (d_input3, d_output3, gradient_h_output3, gradient_v_output3, WIDTH, HEIGHT);

	hipDeviceSynchronize();

	// printf("Launching gradient_horizontal \n");
	gradient_horizontal<< <blocks, threads>> >(d_input5, gradient_h_output5, WIDTH, HEIGHT);
	// printf("Launching gradient_vertical \n");
	gradient_vertical<< <blocks, threads>> >(d_input5, gradient_v_output5, WIDTH, HEIGHT);
	// printf("Launching sobelFilter \n");	
	sobelFilter << <blocks, threads >> > (d_input5, d_output5, gradient_h_output5, gradient_v_output5, WIDTH, HEIGHT);

	hipDeviceSynchronize();
	
	// printf("Copying data back to host \n");
	hipMemcpy(final, d_output, memSize, hipMemcpyDeviceToHost);

	hipMemcpy(final2, d_output2, memSize, hipMemcpyDeviceToHost);

	hipMemcpy(final3, d_output3, memSize, hipMemcpyDeviceToHost);

	hipMemcpy(final4, d_output4, memSize, hipMemcpyDeviceToHost);

	hipMemcpy(final5, d_output5, memSize, hipMemcpyDeviceToHost);

	hipMemcpy(final6, d_output6, memSize, hipMemcpyDeviceToHost);

	hipMemcpy(final7, d_output7, memSize, hipMemcpyDeviceToHost);

	hipMemcpy(final8, d_output8, memSize, hipMemcpyDeviceToHost);

	hipEventRecord(stop_sobel, 0);
  	hipEventSynchronize(stop_sobel);
  	hipEventElapsedTime(&sobel, start_sobel, stop_sobel);

	printf("Total Device Time:  %f s \n", sobel/1000);

	hipError_t err = hipGetLastError();
	if (hipSuccess != err)
	{
		fprintf(stderr, "Cuda error: %s: %s.\n", "Main Loop", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	hipFree(d_input);
	hipFree(d_input);
	hipFree(d_output);
	hipFree(gradient_h_output);
	hipFree(gradient_v_output);

	hipFree(d_input2);
	hipFree(d_input2);
	hipFree(d_output2);
	hipFree(gradient_h_output2);
	hipFree(gradient_v_output2);

	hipFree(d_input3);
	hipFree(d_input3);
	hipFree(d_output3);
	hipFree(gradient_h_output3);
	hipFree(gradient_v_output3);

	hipFree(d_input4);
	hipFree(d_input4);
	hipFree(d_output4);
	hipFree(gradient_h_output4);
	hipFree(gradient_v_output4);

	hipFree(d_input5);
	hipFree(d_input5);
	hipFree(d_output5);
	hipFree(gradient_h_output5);
	hipFree(gradient_v_output5);

	hipFree(d_input6);
	hipFree(d_input6);
	hipFree(d_output6);
	hipFree(gradient_h_output6);
	hipFree(gradient_v_output6);

	hipFree(d_input7);
	hipFree(d_input7);
	hipFree(d_output7);
	hipFree(gradient_h_output7);
	hipFree(gradient_v_output7);

	hipFree(d_input8);
	hipFree(d_input8);
	hipFree(d_output8);
	hipFree(gradient_h_output8);
	hipFree(gradient_v_output8);

 
  hipEventRecord(stop_total, 0);
  hipEventSynchronize(stop_total);
  hipEventElapsedTime(&total, start_total, stop_total);

  printf("Total Time:  %f s \n", total/1000);
  
  // write image
  pgmwrite("../images/image-output_g_apollonian_gasket.ascii.pgm", (void *)final,width, height);
  pgmwrite("../images/image-output_g2_apollonian_gasket.ascii.pgm", (void *)final2,width, height);
  pgmwrite("../images/image-output_g3_apollonian_gasket.ascii.pgm", (void *)final3,width, height);
  pgmwrite("../images/image-output_g4_apollonian_gasket.ascii.pgm", (void *)final4,width, height);
  pgmwrite("../images/image-output_g5_apollonian_gasket.ascii.pgm", (void *)final5,width, height);
  pgmwrite("../images/image-output_g6_apollonian_gasket.ascii.pgm", (void *)final6,width, height);
  pgmwrite("../images/image-output_g7_apollonian_gasket.ascii.pgm", (void *)final7,width, height);
  pgmwrite("../images/image-output_g8_apollonian_gasket.ascii.pgm", (void *)final8,width, height);
    
    
	hipDeviceReset();
	
	return 0;
}

